﻿// -----------------------------------------------------------------------------------------
// NVEnc by rigaya
// -----------------------------------------------------------------------------------------
//
// The MIT License
//
// Copyright (c) 2014-2016 rigaya
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//
// ------------------------------------------------------------------------------------------
#include <array>
#include "ConvertCsp.h"
#include "NVEncFilterDenoisePmd.h"
#include "NVEncParam.h"
#pragma warning (push)
#pragma warning (disable: 4819)
#include "hip/hip_runtime.h"
#include ""
#pragma warning (pop)

#define SRC_TEXTURE g_texImageU16Src
#define GRF_TEXTURE g_texImageU16Grf

texture<uint16_t, hipTextureType2D, hipReadModeElementType> SRC_TEXTURE;
texture<uint16_t, hipTextureType2D, hipReadModeElementType> GRF_TEXTURE;

#include "NVEncFilterDenoisePmd.cuh"

hipError_t denoise_yv12_u10(FrameInfo *pOutputFrame[2], FrameInfo *pGauss, const FrameInfo *pInputFrame,
    int loop_count, const float strength, const float threshold) {
    return denoise_yv12<uint16_t, 10, false>(pOutputFrame, pGauss, pInputFrame, loop_count, strength, threshold);
}

hipError_t denoise_yv12_u10_exp(FrameInfo *pOutputFrame[2], FrameInfo *pGauss, const FrameInfo *pInputFrame,
    int loop_count, const float strength, const float threshold) {
    return denoise_yv12<uint16_t, 10, true>(pOutputFrame, pGauss, pInputFrame, loop_count, strength, threshold);
}

hipError_t denoise_yuv444_u10(FrameInfo *pOutputFrame[2], FrameInfo *pGauss, const FrameInfo *pInputFrame,
    int loop_count, const float strength, const float threshold) {
    return denoise_yuv444<uint16_t, 10, false>(pOutputFrame, pGauss, pInputFrame, loop_count, strength, threshold);
}

hipError_t denoise_yuv444_u10_exp(FrameInfo *pOutputFrame[2], FrameInfo *pGauss, const FrameInfo *pInputFrame,
    int loop_count, const float strength, const float threshold) {
    return denoise_yuv444<uint16_t, 10, true>(pOutputFrame, pGauss, pInputFrame, loop_count, strength, threshold);
}

hipError_t denoise_yv12_u12(FrameInfo *pOutputFrame[2], FrameInfo *pGauss, const FrameInfo *pInputFrame,
    int loop_count, const float strength, const float threshold) {
    return denoise_yv12<uint16_t, 12, false>(pOutputFrame, pGauss, pInputFrame, loop_count, strength, threshold);
}

hipError_t denoise_yv12_u12_exp(FrameInfo *pOutputFrame[2], FrameInfo *pGauss, const FrameInfo *pInputFrame,
    int loop_count, const float strength, const float threshold) {
    return denoise_yv12<uint16_t, 12, true>(pOutputFrame, pGauss, pInputFrame, loop_count, strength, threshold);
}

hipError_t denoise_yuv444_u12(FrameInfo *pOutputFrame[2], FrameInfo *pGauss, const FrameInfo *pInputFrame,
    int loop_count, const float strength, const float threshold) {
    return denoise_yuv444<uint16_t, 12, false>(pOutputFrame, pGauss, pInputFrame, loop_count, strength, threshold);
}

hipError_t denoise_yuv444_u12_exp(FrameInfo *pOutputFrame[2], FrameInfo *pGauss, const FrameInfo *pInputFrame,
    int loop_count, const float strength, const float threshold) {
    return denoise_yuv444<uint16_t, 12, true>(pOutputFrame, pGauss, pInputFrame, loop_count, strength, threshold);
}

hipError_t denoise_yv12_u14(FrameInfo *pOutputFrame[2], FrameInfo *pGauss, const FrameInfo *pInputFrame,
    int loop_count, const float strength, const float threshold) {
    return denoise_yv12<uint16_t, 14, false>(pOutputFrame, pGauss, pInputFrame, loop_count, strength, threshold);
}

hipError_t denoise_yv12_u14_exp(FrameInfo *pOutputFrame[2], FrameInfo *pGauss, const FrameInfo *pInputFrame,
    int loop_count, const float strength, const float threshold) {
    return denoise_yv12<uint16_t, 14, true>(pOutputFrame, pGauss, pInputFrame, loop_count, strength, threshold);
}

hipError_t denoise_yuv444_u14(FrameInfo *pOutputFrame[2], FrameInfo *pGauss, const FrameInfo *pInputFrame,
    int loop_count, const float strength, const float threshold) {
    return denoise_yuv444<uint16_t, 14, false>(pOutputFrame, pGauss, pInputFrame, loop_count, strength, threshold);
}

hipError_t denoise_yuv444_u14_exp(FrameInfo *pOutputFrame[2], FrameInfo *pGauss, const FrameInfo *pInputFrame,
    int loop_count, const float strength, const float threshold) {
    return denoise_yuv444<uint16_t, 14, true>(pOutputFrame, pGauss, pInputFrame, loop_count, strength, threshold);
}

hipError_t denoise_yv12_u16(FrameInfo *pOutputFrame[2], FrameInfo *pGauss, const FrameInfo *pInputFrame,
    int loop_count, const float strength, const float threshold) {
    return denoise_yv12<uint16_t, 16, false>(pOutputFrame, pGauss, pInputFrame, loop_count, strength, threshold);
}

hipError_t denoise_yv12_u16_exp(FrameInfo *pOutputFrame[2], FrameInfo *pGauss, const FrameInfo *pInputFrame,
    int loop_count, const float strength, const float threshold) {
    return denoise_yv12<uint16_t, 16, true>(pOutputFrame, pGauss, pInputFrame, loop_count, strength, threshold);
}

hipError_t denoise_yuv444_u16(FrameInfo *pOutputFrame[2], FrameInfo *pGauss, const FrameInfo *pInputFrame,
    int loop_count, const float strength, const float threshold) {
    return denoise_yuv444<uint16_t, 16, false>(pOutputFrame, pGauss, pInputFrame, loop_count, strength, threshold);
}

hipError_t denoise_yuv444_u16_exp(FrameInfo *pOutputFrame[2], FrameInfo *pGauss, const FrameInfo *pInputFrame,
    int loop_count, const float strength, const float threshold) {
    return denoise_yuv444<uint16_t, 16, true>(pOutputFrame, pGauss, pInputFrame, loop_count, strength, threshold);
}

