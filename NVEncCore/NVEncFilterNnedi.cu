#include "hip/hip_runtime.h"
﻿// -----------------------------------------------------------------------------------------
// NVEnc by rigaya
// -----------------------------------------------------------------------------------------
//
// The MIT License
//
// Copyright (c) 2014-2016 rigaya
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//
// ------------------------------------------------------------------------------------------

#include <map>
#include <array>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#define _USE_MATH_DEFINES
#include <cmath>
#include "convert_csp.h"
#include "NVEncFilterNnedi.h"
#include "NVEncParam.h"
#pragma warning (push)
#pragma warning (disable: 4819)
#include "hip/hip_runtime.h"
#if __CUDACC_VER_MAJOR__ >= 10
#include "hip/hip_fp16.h"
#include "cuda_fp16.hpp"
#endif
#include ""
#pragma warning (pop)
#include "rgy_cuda_util.h"

static const int NNEDI_BLOCK_X       = 32;
static const int NNEDI_BLOCK_Y       = 8;

static const int weight0size = 49 * 4 + 5 * 4 + 9 * 4;
static const int weight0sizenew = 4 * 65 + 4 * 5;

static const int TRASNPOSE_BLOCK_DIM = 16;
static const int TRASNPOSE_TILE_DIM  = 64;

template<typename TypePixel4>
__global__ void kernel_transpose_frame(
    uint8_t *__restrict__ pDst,
    const int dstPitch,
    const int dstWidth,  // = srcHeight
    const int dstHeight, // = srcWidth
    const uint8_t *__restrict__ pSrc,
    const int srcPitch
    ) {
    __shared__ decltype(TypePixel4::x) stemp[TRASNPOSE_TILE_DIM][TRASNPOSE_TILE_DIM + 4];
    {
        const int gSrcIdX = blockIdx.y * TRASNPOSE_TILE_DIM + threadIdx.x * 4;
        const int gSrcIdY = blockIdx.x * TRASNPOSE_TILE_DIM + threadIdx.y;
        const int srcWidth = dstHeight;
        const int srcHeight = dstWidth;
        if (gSrcIdX < srcWidth) {
            for (int j = 0; j < TRASNPOSE_TILE_DIM; j++) {
                TypePixel4 val ={ 0, 0, 0, 0 };
                if (gSrcIdY + j < srcWidth) {
                    TypePixel4 *ptr_src = (TypePixel4 *)(pSrc + (gSrcIdY+j) * srcPitch + gSrcIdX * sizeof(TypePixel4));
                    val = ptr_src[0];
                }
                *(TypePixel4 *)&stemp[threadIdx.y+j][threadIdx.x * 4] = val;
            }
        }
    }
    __syncthreads();

    const int gDstIdX = blockIdx.x * TRASNPOSE_TILE_DIM + threadIdx.x * 4;
    const int gDstIdY = blockIdx.y * TRASNPOSE_TILE_DIM + threadIdx.y;
    if (gDstIdX < dstWidth) {
        for (int j = 0; j < TRASNPOSE_TILE_DIM; j++) {
            if (gDstIdY + j < dstHeight) {
                TypePixel4 val;
                val.x = stemp[threadIdx.x * 4 + 0][threadIdx.y+j];
                val.y = stemp[threadIdx.x * 4 + 1][threadIdx.y+j];
                val.z = stemp[threadIdx.x * 4 + 2][threadIdx.y+j];
                val.w = stemp[threadIdx.x * 4 + 3][threadIdx.y+j];
                TypePixel4 *ptr_dst = (TypePixel4 *)(pDst + (gDstIdY+j) * dstPitch + gDstIdX * sizeof(TypePixel4));
                *ptr_dst = val;
            }
        }
    }
};

__device__ __inline__
static float elliott(float val) {
    return val * __frcp_rn(1.0f + fabs(val));
}

__device__ __inline__
static float exp_(float val) {
    return __expf(clamp(val, -80.0f, 80.0f));
}

#define ENABLE_CUDA_FP16_DEVICE (__CUDACC_VER_MAJOR__ >= 10 && __CUDA_ARCH__ >= 530)
#define ENABLE_CUDA_FP16_HOST   (__CUDACC_VER_MAJOR__ >= 10)

//dot_product1で重み(nns)方向のループアンロールを行う
//これにより、一度sharedメモリからレジスタにのせたpixel情報を使いまわすことができる
#define ENABLE_DP1_WEIGHT_LOOP_UNROLL 1

//重み(nns)方向のループアンロール数
//やりすぎると使用レジスタ数が増え、かえって遅くなる
#define WEIGHT_LOOP 4
static_assert(WEIGHT_LOOP <= WARP_SIZE, "WEIGHT_LOOP < WARP_SIZE");

//ENABLE_DP1_WEIGHT_LOOP_UNROLLに対応して通常の重みの並び [nns*2][nnxy]を変更する
//並びは[nns/WEIGHT_LOOP][nnxy][WEIGHT_LOOP][2]
#define ENABLE_DP1_WEIGHT_ARRAY_OPT (1 && ENABLE_DP1_WEIGHT_LOOP_UNROLL)

//shuffle命令を使ったweight係数の分配により高速化する
#define ENABLE_DP1_SHUFFLE_OPT 1

//スレッド内で複数の出力を同時に計算する
#define THREAD_Y_LOOP 4

#define SSRC(x,y) ((y)*ssrc_dim+(x))
#define SWHT_IDX(i,thIdWeight) ((thIdWeight)*sweight_dim+(i))

template<typename TypeSSrc>
__device__ __inline__
void load_texSrc(TypeSSrc *const ptr_src, const int ssrc_dim, hipTextureObject_t texSrc, const int nnx, const int nny, const int nnx_2_m1, const int nny_2, const int thIdX, const int thIdY, const int gIdX, const int gIdY, const int pix_x_per_thread, const int thread_y_loop);

template<>
__device__ __inline__
void load_texSrc<float>(float *const ptr_src, const int ssrc_dim, hipTextureObject_t texSrc, const int nnx, const int nny, const int nnx_2_m1, const int nny_2, const int thIdX, const int thIdY, const int gIdX, const int gIdY, const int pix_x_per_thread, const int thread_y_loop) {
    for (int y = 0; y + thIdY < NNEDI_BLOCK_Y * thread_y_loop + nny; y += NNEDI_BLOCK_Y) {
        for (int x = 0; x + thIdX < ssrc_dim; x += NNEDI_BLOCK_X) {
            const float px = blockIdx.x * NNEDI_BLOCK_X /*blockDim.x*/ * pix_x_per_thread + thIdX + x - nnx_2_m1 + 0.5f;
            const float py = blockIdx.y * NNEDI_BLOCK_Y /*blockDim.y*/ * thread_y_loop + thIdY + y - nny_2 + 0.5f;
            ptr_src[SSRC(x + thIdX, y + thIdY)] = (float)tex2D<float>(texSrc, px, py) * 256.0f;
        }
    }
}
#if ENABLE_CUDA_FP16_HOST
template<>
__device__ __inline__
void load_texSrc<__half2>(__half2 *const ptr_src, const int ssrc_dim, hipTextureObject_t texSrc, const int nnx, const int nny, const int nnx_2_m1, const int nny_2, const int thIdX, const int thIdY, const int gIdX, const int gIdY, const int pix_x_per_thread, const int thread_y_loop) {
#if ENABLE_CUDA_FP16_DEVICE
    for (int y = 0; y + thIdY < NNEDI_BLOCK_Y * thread_y_loop + nny; y += NNEDI_BLOCK_Y) {
        for (int x = 0; x + thIdX < ssrc_dim; x += NNEDI_BLOCK_X) {
            const float px = blockIdx.x * NNEDI_BLOCK_X /*blockDim.x*/ * pix_x_per_thread + thIdX + x - nnx_2_m1 + 0.5f;
            const float py = blockIdx.y * NNEDI_BLOCK_Y /*blockDim.y*/ * thread_y_loop + thIdY + y - nny_2 + 0.5f;
            ptr_src[SSRC(x + thIdX, y + thIdY)] = __floats2half2_rn(
                tex2D<float>(texSrc, px, py),
                tex2D<float>(texSrc, px+1.0f, py));
        }
    }
#endif //#if ENABLE_CUDA_FP16_DEVICE
}
#endif //#if ENABLE_CUDA_FP16_HOST

template<typename TypePixel, int bit_depth>
__device__ __inline__
TypePixel prescreen_flag() {
    return (1<<bit_depth)-1;
}

template<typename TypePixel, typename TypeWeight, bool scale_dummy, bool src_is_frame, int thread_y_loop>
__device__ __inline__
void dot_product0(
    TypeWeight sum[thread_y_loop][WEIGHT_LOOP],
    const TypePixel *const ptr_src, const int ssrc_dim,
    const TypeWeight *const ptr_weight, const int sweight_dim,
    const TypeWeight *__restrict__ weight_offset,
    const int nnx, const int nny, const int thIdX, const int thIdY,
    const int pix_x_per_thread,
    const float mstd[thread_y_loop][4]
) {
    #pragma unroll
    for (int ithy = 0; ithy < thread_y_loop; ithy++) {
        #pragma unroll
        for (int i = 0; i < WEIGHT_LOOP; i++) {
            sum[ithy][i] = 0.0f;
        }
    }
    const TypeWeight *ptr_w = ptr_weight;
    for (int y = 0; y < nny; y++) {
        const int src_index = (src_is_frame) ? SSRC(thIdX * pix_x_per_thread, thIdY * thread_y_loop + y) : SSRC(0, thIdY * thread_y_loop * NNEDI_BLOCK_X + thIdX);
        const TypePixel *ptr_s = &ptr_src[src_index];

        for (int x = 0; x < nnx; x++, ptr_w++, ptr_s++) {
            TypePixel s0[thread_y_loop];
            #pragma unroll
            for (int ithy = 0; ithy < thread_y_loop; ithy++) {
                s0[ithy] = ptr_s[(src_is_frame) ? (SSRC(0, ithy)) : (SSRC(0, ithy * NNEDI_BLOCK_X))];
            }
            #pragma unroll
            for (int i = 0; i < WEIGHT_LOOP; i++) {
                TypeWeight w0 = ptr_w[SWHT_IDX(0, i)];
                #pragma unroll
                for (int ithy = 0; ithy < thread_y_loop; ithy++) {
                    sum[ithy][i] += s0[ithy] * w0;
                }
            }
        }
    }

    #pragma unroll
    for (int i = 0; i < WEIGHT_LOOP; i++, weight_offset++) {
        const TypeWeight wo = weight_offset[0];
        #pragma unroll
        for (int ithy = 0; ithy < thread_y_loop; ithy++) {
            const TypeWeight scale = (scale_dummy) ? 1.0f : mstd[ithy][2];
            sum[ithy][i] = sum[ithy][i] * scale + wo;
        }
    }
}

template<typename TypePixel, int bit_depth, typename TypeSSrc, int thread_y_loop>
__device__ __inline__
static TypePixel interp_ret(const TypeSSrc *const ptr_src, const int ssrc_dim,
    const bool flag, const int nnx, const int nny, const int thIdX, const int thIdY, int ithy, const int nnx_2_m1, const int nny_2) {
    TypePixel val = prescreen_flag<TypePixel, bit_depth>();
    if (flag) {
        const float tmp =
            (19.0f/32.0f) * (ptr_src[SSRC(thIdX + nnx_2_m1, thIdY * thread_y_loop + ithy + 1)] + ptr_src[SSRC(thIdX + nnx_2_m1, thIdY * thread_y_loop + ithy + 2)])
            - (3.0f/32.0f) * (ptr_src[SSRC(thIdX + nnx_2_m1, thIdY * thread_y_loop + ithy + 0)] + ptr_src[SSRC(thIdX + nnx_2_m1, thIdY * thread_y_loop + ithy + 3)]);
        val = (TypePixel)clamp(tmp * ((1<<bit_depth) / 256.0f) + 0.5f, 0, (1<<bit_depth)-1);
    }
    return val;
}

template<typename TypePixel4, int bit_depth, typename TypeSSrc, typename TypeWeight, bool prescreen_orig, int thread_y_loop>
__global__ void kernel_comute_network0(
    uint8_t *__restrict__ pDst, //top field / bottom field は考慮済みとする
    const int dstPitch, //1行おきなので通常の2倍の値が入っている
    const int dstWidth,
    const int dstHeight,
    hipTextureObject_t texSrc, //有効フィールドのみのテクスチャ(縦解像度は半分)
    const TypeWeight *__restrict__ weight,
    const NnediTargetField targetField
    ) {
    const int pix_x_per_thread = prescreen_orig ? 1 : 4/*4ピクセル分一度に処理する*/;
    const int nnx = (prescreen_orig) ? 12 : 16;
    const int nny = 4;
    const int nnxy = nnx * nny;
    const int nns = 4;
    const int thIdX      = threadIdx.x; //(サイズ: NNEDI_BLOCK_X)
    const int thIdY      = threadIdx.y; //(サイズ: NNEDI_BLOCK_Y)
    const int gIdX       =(blockIdx.x * NNEDI_BLOCK_X /*blockDim.x*/ + thIdX) * pix_x_per_thread;
    const int gIdY       =(blockIdx.y * NNEDI_BLOCK_Y /*blockDim.y*/ + thIdY) * thread_y_loop; //フィールド単位
    const int stmp_dim = (prescreen_orig) ? 8 : 4;

    //sharedメモリのサイズと使途
    //1.src:    (NNEDI_BLOCK_X + nnx) * (NNEDI_BLOCK_Y * thread_y_loop + nny) * sizeof(ptr_src[0])
    //2.temp:   NNEDI_BLOCK_X * NNEDI_BLOCK_Y * stmp_dim * sizeof(ptr_temp[0])
    __shared__ char shared[
        (NNEDI_BLOCK_X * pix_x_per_thread + nnx) * (NNEDI_BLOCK_Y * thread_y_loop + nny) * sizeof(TypeSSrc) +
        NNEDI_BLOCK_X * NNEDI_BLOCK_Y * thread_y_loop * stmp_dim * sizeof(float)
    ];
    TypeSSrc *const ptr_src = (TypeSSrc *)shared;
    const int ssrc_dim = NNEDI_BLOCK_X * pix_x_per_thread + nnx;

    //input(texture) -> shared
    //textureからpixel情報をsharedメモリにロードする
    //範囲外の折り返し等はtextureでやってくれるのでここでは無視
    const int nnx_2_m1 = (prescreen_orig) ? 5 : 6;
    const int nny_2 = nny / 2 - (targetField == NNEDI_GEN_FIELD_BOTTOM ? 1 : 0);
    load_texSrc<TypeSSrc>(ptr_src, ssrc_dim, texSrc, nnx, nny, nnx_2_m1, nny_2, thIdX, thIdY, gIdX, gIdY, pix_x_per_thread, thread_y_loop);
    __syncthreads();

    float *const ptr_temp = (float *)((char *)ptr_src
        + ((NNEDI_BLOCK_X * pix_x_per_thread) + nnx) * (NNEDI_BLOCK_Y * thread_y_loop + nny) * sizeof(ptr_src[0]));
#define STMP_IDX(i,x,y) ( ((y)*(NNEDI_BLOCK_X)+(x)) * stmp_dim + (i))

    float dummy[thread_y_loop][4];
    const int sweight_dim = nnxy;
    if (prescreen_orig) {
        #pragma unroll
        for (int iw = 0; iw < nns; iw += WEIGHT_LOOP) {
            float sum[thread_y_loop][WEIGHT_LOOP]; //レジスタにのることを期待する
            dot_product0<TypeSSrc, TypeWeight, true, true, thread_y_loop>(sum, ptr_src, ssrc_dim, weight+iw*sweight_dim, /*sweight_dim=*/nnxy, weight+48*4+iw, nnx, nny, thIdX, thIdY, pix_x_per_thread, dummy);
            #pragma unroll
            for (int ithy = 0; ithy < thread_y_loop; ithy++) {
                #pragma unroll
                for (int ithw = 0; ithw < WEIGHT_LOOP; ithw++) {
                    ptr_temp[STMP_IDX(iw+ithw, thIdX, thIdY * thread_y_loop + ithy)] = elliott(sum[ithy][ithw]);
                }
            }
        }
        __syncthreads();

        #pragma unroll
        for (int iw = 0; iw < nns; iw += WEIGHT_LOOP) {
            float sum[thread_y_loop][WEIGHT_LOOP]; //レジスタにのることを期待する
            dot_product0<TypeSSrc, TypeWeight, true, false, thread_y_loop>(sum, ptr_temp, stmp_dim, weight+49*4+iw*4, /*sweight_dim=nnxy=*/4, weight+49*4 + 4*4+iw, /*nnx=*/4, /*nny=*/1, thIdX, thIdY, pix_x_per_thread, dummy);
            #pragma unroll
            for (int ithy = 0; ithy < thread_y_loop; ithy++) {
                #pragma unroll
                for (int ithw = 0; ithw < WEIGHT_LOOP; ithw++) {
                    ptr_temp[STMP_IDX(4+iw+ithw, thIdX, thIdY * thread_y_loop + ithy)] = elliott(sum[ithy][ithw]);
                }
            }
        }
        __syncthreads();

        float ret[thread_y_loop][nns]; //レジスタにのることを期待する
        #pragma unroll
        for (int iw = 0; iw < nns; iw += WEIGHT_LOOP) {
            float sum[thread_y_loop][WEIGHT_LOOP]; //レジスタにのることを期待する
            dot_product0<TypeSSrc, TypeWeight, true, false, thread_y_loop>(sum, ptr_temp, stmp_dim, weight + 4*49 + 4*5+iw*8, /*sweight_dim=nnxy=*/8, weight + 4*49 + 4*5 + 4*8+iw, /*nnx=*/8, /*nny=*/1, thIdX, thIdY, pix_x_per_thread, dummy);
            #pragma unroll
            for (int ithy = 0; ithy < thread_y_loop; ithy++) {
                #pragma unroll
                for (int ithw = 0; ithw < WEIGHT_LOOP; ithw++) {
                    ret[ithy][ithw+iw] = sum[ithy][ithw];
                }
            }
        }

        if (gIdX < dstWidth) {
            #pragma unroll
            for (int ithy = 0; ithy < thread_y_loop; ithy++) {
                if ((gIdY + ithy) * 2 < dstHeight) { //縦方向は1行おきの処理となるので "*2"
                    const bool flag = (fmaxf(ret[ithy][2], ret[ithy][3]) <= fmaxf(ret[ithy][0], ret[ithy][1])) ? true : false;
                    decltype(TypePixel4::x) *const ptr_dst = (decltype(TypePixel4::x) *)((uint8_t *)pDst + (gIdY + ithy) * dstPitch + gIdX * sizeof(TypePixel4::x));
                    ptr_dst[0] = interp_ret<decltype(TypePixel4::x), bit_depth, TypeSSrc, thread_y_loop>(ptr_src, ssrc_dim, flag, nnx, nny, thIdX, thIdY, ithy, nnx_2_m1, nny_2);
                }
            }
        }
    } else {
        #pragma unroll
        for (int iw = 0; iw < nns; iw += WEIGHT_LOOP) {
            float sum[thread_y_loop][WEIGHT_LOOP]; //レジスタにのることを期待する
            dot_product0<TypeSSrc, TypeWeight, true, true, thread_y_loop>(sum, ptr_src, ssrc_dim, weight+iw*sweight_dim, /*sweight_dim=*/nnxy, weight+64*4+iw, nnx, nny, thIdX, thIdY, pix_x_per_thread, dummy);
            #pragma unroll
            for (int ithy = 0; ithy < thread_y_loop; ithy++) {
                #pragma unroll
                for (int ithw = 0; ithw < WEIGHT_LOOP; ithw++) {
                    ptr_temp[STMP_IDX(iw+ithw, thIdX, thIdY * thread_y_loop + ithy)] = elliott(sum[ithy][ithw]);
                }
            }
        }
        __syncthreads();

        float ret[thread_y_loop][nns]; //レジスタにのることを期待する
        #pragma unroll
        for (int iw = 0; iw < nns; iw += WEIGHT_LOOP) {
            float sum[thread_y_loop][WEIGHT_LOOP]; //レジスタにのることを期待する
            dot_product0<TypeSSrc, TypeWeight, true, false, thread_y_loop>(sum, ptr_temp, stmp_dim, weight+65*4+iw*4, /*sweight_dim=nnxy=*/4, weight+65*4 + 4*4 + iw, /*nnx=*/4, /*nny=*/1, thIdX, thIdY, pix_x_per_thread, dummy);
            #pragma unroll
            for (int ithy = 0; ithy < thread_y_loop; ithy++) {
                #pragma unroll
                for (int ithw = 0; ithw < WEIGHT_LOOP; ithw++) {
                    ret[ithy][ithw+iw] = sum[ithy][ithw];
                }
            }
        }

        if (gIdX < dstWidth) {
            #pragma unroll
            for (int ithy = 0; ithy < thread_y_loop; ithy++) {
                if ((gIdY + ithy) * 2 < dstHeight) { //縦方向は1行おきの処理となるので "*2"
                    TypePixel4 *const ptr_dst = (TypePixel4 *)((uint8_t *)pDst + (gIdY + ithy) * dstPitch + gIdX * sizeof(decltype(TypePixel4::x)));
                    //1スレッドで4pixel分出力する
                    TypePixel4 out;
                    out.x = interp_ret<decltype(TypePixel4::x), bit_depth, TypeSSrc, thread_y_loop>(ptr_src+0, ssrc_dim, ret[ithy][0] > 0.0f, nnx, nny, thIdX * pix_x_per_thread, thIdY, ithy, nnx_2_m1, nny_2);
                    out.y = interp_ret<decltype(TypePixel4::x), bit_depth, TypeSSrc, thread_y_loop>(ptr_src+1, ssrc_dim, ret[ithy][1] > 0.0f, nnx, nny, thIdX * pix_x_per_thread, thIdY, ithy, nnx_2_m1, nny_2);
                    out.z = interp_ret<decltype(TypePixel4::x), bit_depth, TypeSSrc, thread_y_loop>(ptr_src+2, ssrc_dim, ret[ithy][2] > 0.0f, nnx, nny, thIdX * pix_x_per_thread, thIdY, ithy, nnx_2_m1, nny_2);
                    out.w = interp_ret<decltype(TypePixel4::x), bit_depth, TypeSSrc, thread_y_loop>(ptr_src+3, ssrc_dim, ret[ithy][3] > 0.0f, nnx, nny, thIdX * pix_x_per_thread, thIdY, ithy, nnx_2_m1, nny_2);
                    ptr_dst[0] = out;
                }
            }
        }
    }
}

template<typename T> __device__ __inline__ T setval(float val);
template<> __device__ __inline__ float setval(float val) { return val; };
template<typename TypeSSrc> __device__ __inline__ int kernel_comute_network1_calc_scale_step();
template<> __device__ __inline__ int kernel_comute_network1_calc_scale_step<float>() { return 1; };

template<typename TypeSSrc> __device__
    void kernel_comute_network1_calc_scale_get_sum_sumsq(float& sum, float& sumsq, TypeSSrc tsum, TypeSSrc tsumsq);
template<> __device__ __inline__
    void kernel_comute_network1_calc_scale_get_sum_sumsq<float>(float& sum, float& sumsq, float tsum, float tsumsq) {
    sum = tsum, sumsq = tsumsq;
}
#if ENABLE_CUDA_FP16_HOST
template<> __device__ __inline__ int kernel_comute_network1_calc_scale_step<__half2>() { return 2; };
template<> __device__ __inline__ __half2 setval(float val) { return __float2half2_rn(val); }
template<> __device__ __inline__
    void kernel_comute_network1_calc_scale_get_sum_sumsq<__half2>(float& sum, float& sumsq, __half2 tsum, __half2 tsumsq) {
    sum = ((float)tsum.x + (float)tsum.y) * 256.0f;
    sumsq = ((float)tsumsq.x + (float)tsumsq.y) * 256.0f * 256.0f;
}
#endif //#if ENABLE_CUDA_FP16_HOST

template<typename TypeSSrc, typename TypeWeight>
__device__ __inline__
void kernel_comute_network1_calc_scale(
    float mstd[THREAD_Y_LOOP][4],
    TypeWeight *__restrict__ const ptr_temp,
    const TypeSSrc *__restrict__ const ptr_src, const int ssrc_dim,
    const int nnx, const int nny, const int nnxy,
    const int thIdX, const int thIdY) {
    const int step = kernel_comute_network1_calc_scale_step<TypeSSrc>();
#define TMP_IDX(x,y,i) ((((i)*(nny + NNEDI_BLOCK_Y * THREAD_Y_LOOP)+(y))*NNEDI_BLOCK_X)+(x))
    for (int y = 0; y + thIdY < nny + NNEDI_BLOCK_Y * THREAD_Y_LOOP; y += NNEDI_BLOCK_Y) {
        TypeSSrc sum = setval<TypeSSrc>(0.0f), sumsq = setval<TypeSSrc>(0.0f);
        //まず各ピクセルごとに、x方向の総和をとる
        #pragma unroll (4)
        for (int x = 0; x < nnx; x += step) {
            const auto value = ptr_src[SSRC(x + thIdX, y + thIdY)];
            sum += value;
            sumsq += value * value;
        }
        //一度sharedメモリに格納
        ptr_temp[TMP_IDX(thIdX, thIdY+y, 0)] = sum;
        ptr_temp[TMP_IDX(thIdX, thIdY+y, 1)] = sumsq;
    }
    __syncthreads();

    const float inv_nnxy = __frcp_rn(nnxy);

    //次にy方向の総和をとる
    #pragma unroll
    for (int ithy = 0; ithy < THREAD_Y_LOOP; ithy++) {
        TypeSSrc tsum = setval<TypeSSrc>(0.0f), tsumsq = setval<TypeSSrc>(0.0f);
        #pragma unroll
        for (int y = 0; y < nny; y++) {
            tsum   += ptr_temp[TMP_IDX(thIdX, thIdY*THREAD_Y_LOOP+ithy+y, 0)];
            tsumsq += ptr_temp[TMP_IDX(thIdX, thIdY*THREAD_Y_LOOP+ithy+y, 1)];
        }

        float sum, sumsq;
        kernel_comute_network1_calc_scale_get_sum_sumsq<TypeSSrc>(sum, sumsq, tsum, tsumsq);

        mstd[ithy][3] = 0.0f;
        mstd[ithy][0] = sum * inv_nnxy;
        float tmp = sumsq * inv_nnxy - mstd[ithy][0] * mstd[ithy][0];
        //if (thIdX == 0 && thIdY == 0 && blockIdx.x == 2 && blockIdx.y == 2) {
        //    printf("%e, %e, %e, %e, %e\n", inv_nnxy, tmp, sum, sumsq, mstd[ithy][0]);
        //}
        if (tmp <= FLT_EPSILON) {
            mstd[ithy][1] = 0.0f;
            mstd[ithy][2] = 0.0f;
        } else {
            mstd[ithy][1] = __fsqrt_rn(tmp);
            mstd[ithy][2] = __frcp_rn(mstd[ithy][1]);
        }
    }
#undef TMP_IDX
}
#if ENABLE_CUDA_FP16_HOST && (!ENABLE_CUDA_FP16_DEVICE)
template<>
__device__ __inline__
void kernel_comute_network1_calc_scale(
    float mstd[THREAD_Y_LOOP][4],
    __half2 *__restrict__ const ptr_temp,
    const __half2 *__restrict__ const ptr_src, const int ssrc_dim,
    const int nnx, const int nny, const int nnxy,
    const int thIdX, const int thIdY) {
    //ダミー
}
#endif //#if ENABLE_CUDA_FP16_HOST && (!ENABLE_CUDA_FP16_DEVICE)

template<typename TypeSSrc, typename TypeWeight>
__device__ __inline__
void dot_product_frame1_fp32(
    float sum0[THREAD_Y_LOOP][WEIGHT_LOOP], //レジスタにのることを期待する
    float sum1[THREAD_Y_LOOP][WEIGHT_LOOP], //レジスタにのることを期待する
    TypeSSrc *__restrict__ const ptr_src, const int ssrc_dim,
    const TypeWeight *__restrict__ const ptr_weight, const int sweight_dim,
    const TypeWeight *__restrict__ weight_offset,
    const int nnx, const int nny, const int nns, const int thIdX, const int thIdY,
    const float mstd[THREAD_Y_LOOP][4]
) {
    #pragma unroll
    for (int ithy = 0; ithy < THREAD_Y_LOOP; ithy++) {
        #pragma unroll
        for (int i = 0; i < WEIGHT_LOOP; i++) {
            sum0[ithy][i] = sum1[ithy][i] = 0.0f;
        }
    }
    const TypeWeight *ptr_w = ptr_weight;
    for (int y = 0; y < nny; y++) {
        const TypeSSrc *ptr_s = &ptr_src[SSRC(thIdX, thIdY * THREAD_Y_LOOP + y)];
#if ENABLE_DP1_WEIGHT_ARRAY_OPT
        //#pragma unroll (4)
        for (int x = 0; x < nnx; x++, ptr_s++) {
            //このsharedメモリからロードしたpixelデータをレジスタ上で使いまわすのが重要
            TypeSSrc s0[THREAD_Y_LOOP];
            #pragma unroll
            for (int ithy = 0; ithy < THREAD_Y_LOOP; ithy++) {
                s0[ithy] = ptr_s[SSRC(0, ithy)];
            }
#if ENABLE_DP1_SHUFFLE_OPT
            TypeWeight w;
            if (thIdX < WEIGHT_LOOP*2) w = ptr_w[thIdX];
            ptr_w += WEIGHT_LOOP*2;
            #pragma unroll
            for (int i = 0; i < WEIGHT_LOOP; i++) {
                const auto w0 = __shfl(w, i*2+0);
                const auto w1 = __shfl(w, i*2+1);
                #pragma unroll
                for (int ithy = 0; ithy < THREAD_Y_LOOP; ithy++) {
                    sum0[ithy][i] += s0[ithy] * w0;
                    sum1[ithy][i] += s0[ithy] * w1;
                }
            }
#else
            #pragma unroll
            for (int i = 0; i < WEIGHT_LOOP; i++, ptr_w += 2) {
                const auto w0 = ptr_w[0];
                const auto w1 = ptr_w[1];
                #pragma unroll
                for (int ithy = 0; ithy < THREAD_Y_LOOP; ithy++) {
                    sum0[i][ithy] += s0[ithy] * w0;
                    sum1[i][ithy] += s0[ithy] * w1;
                }
            }
#endif
        }
    }
#else
    #pragma unroll (4)
    for (int x = 0; x < nnx; x++, ptr_w++, ptr_s++) {
        //このsharedメモリからロードしたpixelデータをレジスタ上で使いまわすのが重要
        TypePixel s0[THREAD_Y_LOOP];
        #pragma unroll
        for (int ithy = 0; ithy < THREAD_Y_LOOP; ithy++) {
            s0[ithy] = ptr_s[SSRC(0, ithy*NNEDI_BLOCK_Y)];
        }
        #pragma unroll
        for (int i = 0; i < WEIGHT_LOOP; i++) {
            TypeWeight w0 = ptr_w[SWHT_IDX(0, i)];
            TypeWeight w1 = ptr_w[SWHT_IDX(0, i+nns)];
            #pragma unroll
            for (int ithy = 0; ithy < THREAD_Y_LOOP; ithy++) {
                sum0[i][ithy] += s0[ithy] * w0;
                sum1[i][ithy] += s0[ithy] * w1;
            }
        }
    }
#endif
#if ENABLE_DP1_WEIGHT_ARRAY_OPT
    #pragma unroll
    for (int i = 0; i < WEIGHT_LOOP; i++, weight_offset += 2) {
        #pragma unroll
        for (int ithy = 0; ithy < THREAD_Y_LOOP; ithy++) {
            sum0[ithy][i] = sum0[ithy][i] * mstd[ithy][2] + weight_offset[0];
            sum1[ithy][i] = sum1[ithy][i] * mstd[ithy][2] + weight_offset[1];
        }
    }
#else
    #pragma unroll
    for (int i = 0; i < WEIGHT_LOOP; i++, weight_offset++) {
        #pragma unroll
        for (int ithy = 0; ithy < THREAD_Y_LOOP; ithy++) {
            sum0[ithy][i] = sum0[ithy][i] * mstd[ithy][2] + weight_offset[0];
            sum1[ithy][i] = sum1[ithy][i] * mstd[ithy][2] + weight_offset[nns];
        }
    }
#endif
}

#if ENABLE_CUDA_FP16_HOST
__device__ __inline__
void dot_product_frame1_fp16(
    __half2 sum[THREAD_Y_LOOP][WEIGHT_LOOP],
    __half2 *__restrict__ const ptr_src, const int ssrc_dim,
    const __half2 *__restrict__ const ptr_weight, const int sweight_dim,
    const __half2 *__restrict__ weight_offset,
    const int nnx, const int nny, const int nns, const int thIdX, const int thIdY,
    const __half2 weight_scale[THREAD_Y_LOOP]
) {
#if ENABLE_CUDA_FP16_DEVICE
    #pragma unroll
    for (int ithy = 0; ithy < THREAD_Y_LOOP; ithy++) {
        #pragma unroll
        for (int i = 0; i < WEIGHT_LOOP; i++) {
            sum[ithy][i] = setval<__half2>(0.0f);
        }
    }
    const __half2 *ptr_w = ptr_weight;
    for (int y = 0; y < nny; y++) {
        const __half2 *ptr_s = &ptr_src[SSRC(thIdX, thIdY * THREAD_Y_LOOP + y)];
        //#pragma unroll (4)
        for (int x = 0; x < nnx; x += 2, ptr_s += 2) {
            //このsharedメモリからロードしたpixelデータをレジスタ上で使いまわすのが重要
            __half2 s0[THREAD_Y_LOOP];
            #pragma unroll
            for (int ithy = 0; ithy < THREAD_Y_LOOP; ithy++) {
                s0[ithy] = ptr_s[SSRC(0, ithy)];
            }
            //[nns/WEIGHT_LOOP][nnxy][WEIGHT_LOOP][2]
            __half2 w;
            if (thIdX < WEIGHT_LOOP*2) w = ptr_w[thIdX];
            ptr_w += WEIGHT_LOOP*2;
            #pragma unroll
            for (int i = 0; i < WEIGHT_LOOP; i++) {
                __half2 w0 = __shfl(w,            +i);
                __half2 w1 = __shfl(w, WEIGHT_LOOP+i);
                #pragma unroll
                for (int ithy = 0; ithy < THREAD_Y_LOOP; ithy++) {
                    sum[ithy][i] += __low2half2(s0[ithy]) * w0;
                    sum[ithy][i] += __high2half2(s0[ithy]) * w1;
                }
            }
        }
    }
    #pragma unroll
    for (int i = 0; i < WEIGHT_LOOP; i++, weight_offset++) {
        #pragma unroll
        for (int ithy = 0; ithy < THREAD_Y_LOOP; ithy++) {
            sum[ithy][i] = sum[ithy][i] * weight_scale[ithy] + weight_offset[0];
        }
    }
#endif //#if ENABLE_CUDA_FP16_DEVICE
}
#endif //#if ENABLE_CUDA_FP16_HOST

__device__ __inline__
void kernel_comute_network1_dot_product(
    float wsum[THREAD_Y_LOOP],
    float vsum[THREAD_Y_LOOP],
    float *const ptr_src, const int ssrc_dim,
    const float *const weight,
    float mstd[THREAD_Y_LOOP][4],
    const int nnx, const int nny, const int nnxy, const int nns,
    const int thIdX, const int thIdY) {
    const int sweight_dim = (ENABLE_DP1_WEIGHT_ARRAY_OPT) ? 2 * nnxy : nnxy;
    for (int iw = 0; iw < nns; iw += WEIGHT_LOOP) {
        float sum0[THREAD_Y_LOOP][WEIGHT_LOOP]; //レジスタにのることを期待する
        dot_product0<float, float, false, true, THREAD_Y_LOOP>(sum0, ptr_src, ssrc_dim, weight+ (iw)*nnxy, sweight_dim, weight + (nns*2)*nnxy + iw, nnx, nny, thIdX, thIdY, 1, mstd);

        float sum1[THREAD_Y_LOOP][WEIGHT_LOOP]; //レジスタにのることを期待する
        dot_product0<float, float, false, true, THREAD_Y_LOOP>(sum1, ptr_src, ssrc_dim, weight+ (nns+iw)*nnxy, sweight_dim, weight + (nns*2)*nnxy+nns + iw, nnx, nny, thIdX, thIdY, 1, mstd);

        #pragma unroll
        for (int ithy = 0; ithy < THREAD_Y_LOOP; ithy++) {
            #pragma unroll
            for (int ithw = 0; ithw < WEIGHT_LOOP; ithw++) {
                float ret0 = exp_(sum0[ithy][ithw]);
                float ret1 = sum1[ithy][ithw];
                wsum[ithy] += ret0;
                vsum[ithy] += ret0 * (ret1 * __frcp_rn(1.0f + fabs(ret1)));
            }
        }
    }
}

#if ENABLE_CUDA_FP16_HOST
__device__ __inline__
void kernel_comute_network1_dot_product(
    float wsum[THREAD_Y_LOOP],
    float vsum[THREAD_Y_LOOP],
    __half2 *const ptr_src, const int ssrc_dim,
    const __half2 *const weight,
    float mstd[THREAD_Y_LOOP][4],
    const int nnx, const int nny, const int nnxy, const int nns,
    const int thIdX, const int thIdY) {
    //未実装
    assert(0);
}
#endif //#if ENABLE_CUDA_FP16_HOST

__device__ __inline__
void kernel_comute_network1_dot_product_opt(
    float wsum[THREAD_Y_LOOP],
    float vsum[THREAD_Y_LOOP],
    float *const ptr_src, const int ssrc_dim,
    const float *const weight,
    float mstd[THREAD_Y_LOOP][4],
    const int nnx, const int nny, const int nnxy, const int nns,
    const int thIdX, const int thIdY) {
    const int sweight_dim = (ENABLE_DP1_WEIGHT_ARRAY_OPT) ? 2 * nnxy : nnxy;
    for (int iw = 0; iw < nns; iw += WEIGHT_LOOP) {
        float sum0[THREAD_Y_LOOP][WEIGHT_LOOP]; //レジスタにのることを期待する
        float sum1[THREAD_Y_LOOP][WEIGHT_LOOP]; //レジスタにのることを期待する
        // 重み(nns)方向に、WEIGHT_LOOP分のdotproduct
        // sum0[i] <- iw     - iw+WEIGHT_LOOP
        // sum1[i] <- iw+nns - iw+WEIGHT_LOOP+nns
        dot_product_frame1_fp32(sum0, sum1, ptr_src, ssrc_dim, weight+iw*sweight_dim, sweight_dim, weight + (nns*2)*nnxy + iw*2, nnx, nny, nns, thIdX, thIdY, mstd);
        #pragma unroll
        for (int ithy = 0; ithy < THREAD_Y_LOOP; ithy++) {
            #pragma unroll
            for (int ithw = 0; ithw < WEIGHT_LOOP; ithw++) {
                float ret0 = exp_(sum0[ithy][ithw]);
                float ret1 = sum1[ithy][ithw];
                wsum[ithy] += ret0;
                vsum[ithy] += ret0 * (ret1 * __frcp_rn(1.0f + fabs(ret1)));
            }
        }
    }
}

#if ENABLE_CUDA_FP16_HOST
__device__ __inline__
void kernel_comute_network1_dot_product_opt(
    float wsum[THREAD_Y_LOOP],
    float vsum[THREAD_Y_LOOP],
    __half2 *const ptr_src, const int ssrc_dim,
    const __half2 *const weight,
    float mstd[THREAD_Y_LOOP][4],
    const int nnx, const int nny, const int nnxy, const int nns,
    const int thIdX, const int thIdY) {
#if ENABLE_CUDA_FP16_DEVICE
    const int sweight_dim = nnxy;
    for (int iw = 0; iw < nns; iw += WEIGHT_LOOP) {
        __half2 sum[THREAD_Y_LOOP][WEIGHT_LOOP]; //レジスタにのることを期待する
        // 重み(nns)方向に、WEIGHT_LOOP分のdotproduct
        // sum0[i] <- iw     - iw+WEIGHT_LOOP
        // sum1[i] <- iw+nns - iw+WEIGHT_LOOP+nns
        __half2 weight_scale[THREAD_Y_LOOP];
        #pragma unroll
        for (int ithy = 0; ithy < THREAD_Y_LOOP; ithy++) {
            weight_scale[ithy] = __float2half2_rn(mstd[ithy][2] * 256.0f); // *weight[nns*(nnxy+1)] <<<<<<<< scalingはとりあえずなしで
        }
        dot_product_frame1_fp16(sum, ptr_src, ssrc_dim, weight+iw*sweight_dim, sweight_dim, weight + nns*nnxy + iw, nnx, nny, nns, thIdX, thIdY, weight_scale);
        #pragma unroll
        for (int ithy = 0; ithy < THREAD_Y_LOOP; ithy++) {
            #pragma unroll
            for (int ithw = 0; ithw < WEIGHT_LOOP; ithw++) {
                float ret0 = exp_(__low2float(sum[ithy][ithw]));
                float ret1 = __high2float(sum[ithy][ithw]);
                wsum[ithy] += ret0;
                vsum[ithy] += ret0 * (ret1 * __frcp_rn(1.0f + fabs(ret1)));
            }
        }
    }
#endif //#if ENABLE_CUDA_FP16_DEVICE
}
#endif //#if ENABLE_CUDA_FP16_HOST


template<typename TypePixel, int bit_depth, typename TypeSSrc, typename TypeWeight, int nnx, int nny>
__global__ void kernel_comute_network1(
    uint8_t *__restrict__ pDst, //top field / bottom field は考慮済みとする
    const int dstPitch, //1行おきなので通常の2倍の値が入っている
    const int dstWidth,
    const int dstHeight,
    hipTextureObject_t texSrc, //有効フィールドのみのテクスチャ(縦解像度は半分)
    const TypeWeight *__restrict__ weight10,
    const TypeWeight *__restrict__ weight11,
    const int nns,  // len = nns*2
    const int quals,
    const NnediTargetField targetField,
    bool prescreen
) {
    const int thIdX      = threadIdx.x; //(サイズ: NNEDI_BLOCK_X)
    const int thIdY      = threadIdx.y; //(サイズ: NNEDI_BLOCK_Y)
    const int gIdX       = blockIdx.x * NNEDI_BLOCK_X /*blockDim.x*/ + thIdX;
    const int gIdY       =(blockIdx.y * NNEDI_BLOCK_Y /*blockDim.y*/ + thIdY) * THREAD_Y_LOOP; //フィールド単位
    const int nnxy       = nnx * nny;

    //sharedメモリのサイズと使途
    //1.src: (NNEDI_BLOCK_X + nnx) * (NNEDI_BLOCK_Y * THREAD_Y_LOOP + nny) * sizeof(ptr_src[0])
    //2.tmp: (nny + NNEDI_BLOCK_Y * THREAD_Y_LOOP) * NNEDI_BLOCK_X * 2 * sizeof(ptr_temp[0])
    alignas(128) extern __shared__ char shared[];
    TypeSSrc *const ptr_src = (TypeSSrc *)shared;
    const int ssrc_dim = NNEDI_BLOCK_X + nnx;

    //input(texture) -> shared
    //textureからpixel情報をsharedメモリにロードする
    //範囲外の折り返し等はtextureでやってくれるのでここでは無視
    const int nnx_2_m1 = nnx / 2 - 1;
    const int nny_2 = nny / 2 - (targetField == NNEDI_GEN_FIELD_BOTTOM ? 1 : 0);
    load_texSrc<TypeSSrc>(ptr_src, ssrc_dim, texSrc, nnx, nny, nnx_2_m1, nny_2, thIdX, thIdY, gIdX, gIdY, 1, THREAD_Y_LOOP);
    __syncthreads();

    TypeWeight *const ptr_temp = (TypeWeight *)((char *)shared
        + (NNEDI_BLOCK_X + nnx) * (NNEDI_BLOCK_Y * THREAD_Y_LOOP + nny) * sizeof(ptr_src[0]));

    float mstd[THREAD_Y_LOOP][4];
    kernel_comute_network1_calc_scale(mstd, ptr_temp, ptr_src, ssrc_dim, nnx, nny, nnxy, thIdX, thIdY);

    uint8_t *const ptr_dst_base = (uint8_t *)pDst + gIdY * dstPitch + gIdX * sizeof(TypePixel);
    uint32_t flag_sum = 0xffffffff; //処理するかどうかのフラグ
    if (prescreen) {
        flag_sum = 0x00;
        uint8_t *ptr_dst = ptr_dst_base;
        #pragma unroll
        for (int ithy = 0; ithy < THREAD_Y_LOOP; ithy++, ptr_dst += dstPitch) {
            uint32_t flag = 0x00;
            if ((gIdY + ithy) * 2 < dstHeight) { //縦方向は1行おきの処理となるので "*2"
                flag = (((TypePixel *)ptr_dst)[0] == prescreen_flag<TypePixel, bit_depth>()) ? 0x01 << ithy : 0x00;
            }
            flag_sum |= flag;
            static_assert(THREAD_Y_LOOP <= sizeof(flag_sum) * 8, "THREAD_Y_LOOP <= sizeof(flag_sum) * 8");
        }
    }

#if 0
                                      |<-------- nns*2 --------->|
                                    WEIGHT_LOOP
                                      |<-->| ---> 繰り返し処理
                                 ---  |--------------------------|
                                      |                          |
                                      |                          |
                                      |                          |
                             nnxy     |                          |
                                      |                          |
                                      |                          |
                                      |                          |
                                 ---  |--------------------------|

                |<----   nnxy  --->|
            --- |------------------|  |----|
NNEDI_BLOCK_X   |                  |  |    | <-- 各スレッドはこの出力の1pixel分(縦方向)をそれぞれ担当
*NNEDI_BLOCK_Y  |                  |  |    |      横: WEIGHT_LOOP
            --- |                  |  |----|      縦: NNEDI_BLOCK_X * NNEDI_BLOCK_Y
                |                  |
                |                  |
        pixels  |                  |
           |    |                  |
           |    |                  |
        　↓    |                  |

#endif
    //weightの先頭のポインタ
    if (__any(flag_sum)) { //どのpixelも処理する必要がなければ、スキップする
        for (int iquality = 0; iquality < quals; iquality++) {
            const TypeWeight *const weight = (iquality) ? weight11 : weight10;
            float wsum[THREAD_Y_LOOP], vsum[THREAD_Y_LOOP];
            #pragma unroll
            for (int ithy = 0; ithy < THREAD_Y_LOOP; ithy++) {
                wsum[ithy] = vsum[ithy] = 0.0f;
            }
            if (ENABLE_DP1_WEIGHT_LOOP_UNROLL) {
                kernel_comute_network1_dot_product_opt(
                    wsum, vsum, ptr_src, ssrc_dim, weight, mstd, nnx, nny, nnxy, nns, thIdX, thIdY);
            } else {
                kernel_comute_network1_dot_product(
                    wsum, vsum, ptr_src, ssrc_dim, weight, mstd, nnx, nny, nnxy, nns, thIdX, thIdY);
            }

            const float min_weight_sum = 1e-10f;
            for (int ithy = 0; ithy < THREAD_Y_LOOP; ithy++) {
                if (wsum[ithy] > min_weight_sum) {
                    mstd[ithy][3] += ((5.0f * vsum[ithy]) * __frcp_rn(wsum[ithy])) * mstd[ithy][1];
                }
                mstd[ithy][3] += mstd[ithy][0];
            }
        }

        if (gIdX < dstWidth) {
            const float scale = (1<<bit_depth) / 256.0f * ((quals > 1) ? 0.5f : 1.0f);
            uint8_t *ptr_dst = (uint8_t *)ptr_dst_base;
            for (int ithy = 0; ithy < THREAD_Y_LOOP; ithy++, ptr_dst += dstPitch) {
                if ((flag_sum & (1<<ithy)) && (gIdY + ithy) * 2 < dstHeight) { //縦方向は1行おきの処理となるので "*2"
                    ((TypePixel *)ptr_dst)[0] = (TypePixel)clamp(mstd[ithy][3] * scale + 0.5f, 0.0f, (1<<bit_depth)-1.0f);
                }
            }
        }
    }
}

template<typename TypePixel>
hipError_t setTexField(hipTextureObject_t& texSrc, const FrameInfo *pFrame, const NnediTargetField targetField) {
    texSrc = 0;

    hipResourceDesc resDescSrc;
    memset(&resDescSrc, 0, sizeof(resDescSrc));
    resDescSrc.resType = hipResourceTypePitch2D;
    resDescSrc.res.pitch2D.desc = hipCreateChannelDesc<TypePixel>();
    resDescSrc.res.pitch2D.pitchInBytes = pFrame->pitch * 2; //1行おきなので通常の2倍
    resDescSrc.res.pitch2D.width = pFrame->width;
    resDescSrc.res.pitch2D.height = pFrame->height / 2; //フィールドなので半分
    resDescSrc.res.pitch2D.devPtr = (uint8_t *)pFrame->ptr
        + (pFrame->pitch * ((targetField == NNEDI_GEN_FIELD_TOP) ? 1 : 0)); //有効なほうのフィールドを選択

    hipTextureDesc texDescSrc;
    memset(&texDescSrc, 0, sizeof(texDescSrc));
    texDescSrc.addressMode[0]   = hipAddressModeClamp;
    texDescSrc.addressMode[1]   = hipAddressModeClamp;
    texDescSrc.filterMode       = hipFilterModePoint;
    texDescSrc.readMode         = hipReadModeNormalizedFloat;
    texDescSrc.normalizedCoords = 0;

    return hipCreateTextureObject(&texSrc, &resDescSrc, &texDescSrc, nullptr);
}

template<typename TypePixel4, int bit_depth, typename TypeSSrc, typename TypeWeight>
hipError_t nnedi_compute_network_0(FrameInfo *pOutputPlane,
    hipTextureObject_t texSrc,
    const TypeWeight *weight0,
    const VppNnediPreScreen pre_screen,
    const NnediTargetField targetField,
    hipStream_t stream
) {
    dim3 blockSize(NNEDI_BLOCK_X, NNEDI_BLOCK_Y);

    auto cudaerr = hipSuccess;
    if (pre_screen == VPP_NNEDI_PRE_SCREEN_ORIGINAL) {
        const int thread_y_loop_org = 2;
        dim3 gridSize(
            divCeil(pOutputPlane->width, blockSize.x),
            divCeil(pOutputPlane->height / 2, blockSize.y * thread_y_loop_org));
        kernel_comute_network0<TypePixel4, bit_depth, TypeSSrc, TypeWeight, true, thread_y_loop_org><<<gridSize, blockSize, 0, stream>>>(
            (uint8_t *)pOutputPlane->ptr + pOutputPlane->pitch * (targetField == NNEDI_GEN_FIELD_TOP ? 0 : 1), //生成するほうのフィールドを選択
            pOutputPlane->pitch * 2, //1行おきなので通常の2倍
            pOutputPlane->width,
            pOutputPlane->height,
            texSrc, //有効フィールドのみのテクスチャ(縦解像度は半分)
            weight0, targetField);
        cudaerr = hipGetLastError();
    } else if (pre_screen >= VPP_NNEDI_PRE_SCREEN_NEW) {
        const int thread_y_loop_new = 2;
        dim3 gridSize(
            divCeil(pOutputPlane->width, blockSize.x * 4 /*4ピクセル分一度に処理する*/),
            divCeil(pOutputPlane->height / 2, blockSize.y * thread_y_loop_new));
        kernel_comute_network0<TypePixel4, bit_depth, TypeSSrc, TypeWeight, false, thread_y_loop_new><<<gridSize, blockSize, 0, stream>>>(
            (uint8_t *)pOutputPlane->ptr + pOutputPlane->pitch * (targetField == NNEDI_GEN_FIELD_TOP ? 0 : 1), //生成するほうのフィールドを選択
            pOutputPlane->pitch * 2, //1行おきなので通常の2倍
            pOutputPlane->width,
            pOutputPlane->height,
            texSrc, //有効フィールドのみのテクスチャ(縦解像度は半分)
            weight0, targetField);
        cudaerr = hipGetLastError();
    } else {
        const auto outputFrameInfoEx = getFrameInfoExtra(pOutputPlane);
        cudaerr = hipMemset2DAsync(
            (uint8_t *)pOutputPlane->ptr + pOutputPlane->pitch * (targetField == NNEDI_GEN_FIELD_TOP ? 0 : 1),
            pOutputPlane->pitch * 2, //1行おきなので通常の2倍
            -1, //value
            outputFrameInfoEx.width_byte,
            pOutputPlane->height / 2,
            stream);
    }
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    return cudaerr;
}

template<typename TypePixel, int bit_depth, typename TypeSSrc, typename TypeWeight>
hipError_t nnedi_compute_network_1(
    FrameInfo *pOutputFrame,
    hipTextureObject_t texSrc,
    const TypeWeight *weight10,
    const TypeWeight *weight11,
    const NnediTargetField targetField,
    const VppNnediNSize nsize,
    const int nns,
    const VppNnediQuality quality,
    const VppNnediPreScreen pre_screen,
    hipStream_t stream
) {
    dim3 blockSize(NNEDI_BLOCK_X, NNEDI_BLOCK_Y);
    dim3 gridSize(
        divCeil(pOutputFrame->width, blockSize.x),
        divCeil(pOutputFrame->height / 2, blockSize.y * THREAD_Y_LOOP));

    const int nnx = NVEncFilterNnedi::sizeNX[nsize];
    const int nny = NVEncFilterNnedi::sizeNY[nsize];
    const int shared_mem_size =
        (NNEDI_BLOCK_X + nnx) * (NNEDI_BLOCK_Y * THREAD_Y_LOOP + nny) * sizeof(TypeSSrc) + //src
        (NNEDI_BLOCK_Y * THREAD_Y_LOOP + nny) * NNEDI_BLOCK_X * 2 * sizeof(TypeWeight); //temp

    switch (nsize) {
    case VPP_NNEDI_NSIZE_8x6:
        kernel_comute_network1<TypePixel, bit_depth, TypeSSrc, TypeWeight, 8, 6><<<gridSize, blockSize, shared_mem_size, stream>>>(
            (uint8_t *)pOutputFrame->ptr + pOutputFrame->pitch * ((targetField == NNEDI_GEN_FIELD_TOP) ? 0 : 1), //生成するほうのフィールドを選択
            pOutputFrame->pitch * 2, //1行おきなので通常の2倍
            pOutputFrame->width,
            pOutputFrame->height,
            texSrc,
            weight10, weight11,
            nns, (int)quality, targetField, pre_screen != VPP_NNEDI_PRE_SCREEN_NONE);
        break;
    case VPP_NNEDI_NSIZE_16x6:
        kernel_comute_network1<TypePixel, bit_depth, TypeSSrc, TypeWeight, 16, 6><<<gridSize, blockSize, shared_mem_size, stream>>>(
            (uint8_t *)pOutputFrame->ptr + pOutputFrame->pitch * ((targetField == NNEDI_GEN_FIELD_TOP) ? 0 : 1), //生成するほうのフィールドを選択
            pOutputFrame->pitch * 2, //1行おきなので通常の2倍
            pOutputFrame->width,
            pOutputFrame->height,
            texSrc,
            weight10, weight11,
            nns, (int)quality, targetField, pre_screen != VPP_NNEDI_PRE_SCREEN_NONE);
        break;
    case VPP_NNEDI_NSIZE_32x6:
        kernel_comute_network1<TypePixel, bit_depth, TypeSSrc, TypeWeight, 32, 6><<<gridSize, blockSize, shared_mem_size, stream>>>(
            (uint8_t *)pOutputFrame->ptr + pOutputFrame->pitch * ((targetField == NNEDI_GEN_FIELD_TOP) ? 0 : 1), //生成するほうのフィールドを選択
            pOutputFrame->pitch * 2, //1行おきなので通常の2倍
            pOutputFrame->width,
            pOutputFrame->height,
            texSrc,
            weight10, weight11,
            nns, (int)quality, targetField, pre_screen != VPP_NNEDI_PRE_SCREEN_NONE);
        break;
    case VPP_NNEDI_NSIZE_48x6:
        kernel_comute_network1<TypePixel, bit_depth, TypeSSrc, TypeWeight, 48, 6><<<gridSize, blockSize, shared_mem_size, stream>>>(
            (uint8_t *)pOutputFrame->ptr + pOutputFrame->pitch * ((targetField == NNEDI_GEN_FIELD_TOP) ? 0 : 1), //生成するほうのフィールドを選択
            pOutputFrame->pitch * 2, //1行おきなので通常の2倍
            pOutputFrame->width,
            pOutputFrame->height,
            texSrc,
            weight10, weight11,
            nns, (int)quality, targetField, pre_screen != VPP_NNEDI_PRE_SCREEN_NONE);
        break;
    case VPP_NNEDI_NSIZE_8x4:
        kernel_comute_network1<TypePixel, bit_depth, TypeSSrc, TypeWeight, 8, 4><<<gridSize, blockSize, shared_mem_size, stream>>>(
            (uint8_t *)pOutputFrame->ptr + pOutputFrame->pitch * ((targetField == NNEDI_GEN_FIELD_TOP) ? 0 : 1), //生成するほうのフィールドを選択
            pOutputFrame->pitch * 2, //1行おきなので通常の2倍
            pOutputFrame->width,
            pOutputFrame->height,
            texSrc,
            weight10, weight11,
            nns, (int)quality, targetField, pre_screen != VPP_NNEDI_PRE_SCREEN_NONE);
        break;
    case VPP_NNEDI_NSIZE_16x4:
        kernel_comute_network1<TypePixel, bit_depth, TypeSSrc, TypeWeight, 16, 4><<<gridSize, blockSize, shared_mem_size, stream>>>(
            (uint8_t *)pOutputFrame->ptr + pOutputFrame->pitch * ((targetField == NNEDI_GEN_FIELD_TOP) ? 0 : 1), //生成するほうのフィールドを選択
            pOutputFrame->pitch * 2, //1行おきなので通常の2倍
            pOutputFrame->width,
            pOutputFrame->height,
            texSrc,
            weight10, weight11,
            nns, (int)quality, targetField, pre_screen != VPP_NNEDI_PRE_SCREEN_NONE);
        break;
    case VPP_NNEDI_NSIZE_32x4:
        kernel_comute_network1<TypePixel, bit_depth, TypeSSrc, TypeWeight, 32, 4><<<gridSize, blockSize, shared_mem_size, stream>>>(
            (uint8_t *)pOutputFrame->ptr + pOutputFrame->pitch * ((targetField == NNEDI_GEN_FIELD_TOP) ? 0 : 1), //生成するほうのフィールドを選択
            pOutputFrame->pitch * 2, //1行おきなので通常の2倍
            pOutputFrame->width,
            pOutputFrame->height,
            texSrc,
            weight10, weight11,
            nns, (int)quality, targetField, pre_screen != VPP_NNEDI_PRE_SCREEN_NONE);
        break;
    default:
        return hipErrorAssert;
    }
    auto cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    return cudaerr;
}

template<typename TypePixel, typename TypePixel4, int bit_depth, typename TypeSSrc, typename TypeWeight>
hipError_t proc_plane(
    FrameInfo *pOutputPlane,
    const FrameInfo *pInputPlane,
    const std::shared_ptr<NVEncFilterParamNnedi> pNnediParam,
    const NnediTargetField targetField,
    const float *weight0,
    const TypeWeight *weight10,
    const TypeWeight *weight11,
    hipStream_t stream
) {
    const auto inputFrameInfoEx = getFrameInfoExtra(pInputPlane);
    // 有効なほうのフィールドをコピー
    auto cudaerr = hipMemcpy2DAsync(
        (uint8_t *)pOutputPlane->ptr + pOutputPlane->pitch * (targetField == NNEDI_GEN_FIELD_TOP ? 1 : 0),
        pOutputPlane->pitch * 2, //1行おきなので通常の2倍
        (uint8_t *)pInputPlane->ptr + pInputPlane->pitch * (targetField == NNEDI_GEN_FIELD_TOP ? 1 : 0),
        pInputPlane->pitch * 2,  //1行おきなので通常の2倍
        inputFrameInfoEx.width_byte,
        pInputPlane->height / 2,
        hipMemcpyDeviceToDevice,
        stream);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }

    hipTextureObject_t texSrc = 0;
    cudaerr = setTexField<TypePixel>(texSrc, pInputPlane, targetField);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    cudaerr = nnedi_compute_network_0<TypePixel4, bit_depth, float, float>(pOutputPlane,
        texSrc,
        weight0,
        pNnediParam->nnedi.pre_screen,
        targetField,
        stream);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    cudaerr = nnedi_compute_network_1<TypePixel, bit_depth, TypeSSrc, TypeWeight>(
        pOutputPlane,
        texSrc,
        weight10,
        weight11,
        targetField,
        pNnediParam->nnedi.nsize,
        pNnediParam->nnedi.nns,
        pNnediParam->nnedi.quality,
        pNnediParam->nnedi.pre_screen,
        stream);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    cudaerr = hipDestroyTextureObject(texSrc);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    return cudaerr;
}

template<typename TypePixel, typename TypePixel4, int bit_depth, typename TypeSSrc, typename TypeWeight>
hipError_t proc_frame(FrameInfo *pOutputFrame,
    const FrameInfo *pInputFrame,
    const std::shared_ptr<NVEncFilterParamNnedi> pNnediParam,
    const NnediTargetField targetField,
    const void *weight0,
    const void *weight10,
    const void *weight11,
    hipStream_t stream
) {
    static_assert(sizeof(TypePixel4) == sizeof(TypePixel) * 4, "sizeof(TypePixel4) == sizeof(TypePixel) * 4");
    hipError_t cudaerr = hipSuccess;
    const auto planeInputY = getPlane(pInputFrame, RGY_PLANE_Y);
    const auto planeInputU = getPlane(pInputFrame, RGY_PLANE_U);
    const auto planeInputV = getPlane(pInputFrame, RGY_PLANE_V);
    auto planeOutputY = getPlane(pOutputFrame, RGY_PLANE_Y);
    auto planeOutputU = getPlane(pOutputFrame, RGY_PLANE_U);
    auto planeOutputV = getPlane(pOutputFrame, RGY_PLANE_V);

    cudaerr = proc_plane<TypePixel, TypePixel4, bit_depth, TypeSSrc, TypeWeight>(&planeOutputY, &planeInputY, pNnediParam, targetField, (const float *)weight0, (const TypeWeight *)weight10, (const TypeWeight *)weight11, stream);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    cudaerr = proc_plane<TypePixel, TypePixel4, bit_depth, TypeSSrc, TypeWeight>(&planeOutputU, &planeInputU, pNnediParam, targetField, (const float *)weight0, (const TypeWeight *)weight10, (const TypeWeight *)weight11, stream);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    cudaerr = proc_plane<TypePixel, TypePixel4, bit_depth, TypeSSrc, TypeWeight>(&planeOutputV, &planeInputV, pNnediParam, targetField, (const float *)weight0, (const TypeWeight *)weight10, (const TypeWeight *)weight11, stream);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    return cudaerr;
}

const int NVEncFilterNnedi::sizeNX[] = { 8, 16, 32, 48, 8, 16, 32 };
const int NVEncFilterNnedi::sizeNY[] = { 6, 6, 6, 6, 4, 4, 4 };
const int NVEncFilterNnedi::sizeNN[] = { 16, 32, 64, 128, 256 };

NVEncFilterNnedi::NVEncFilterNnedi() : m_weight0(), m_weight1() {
    m_sFilterName = _T("nnedi");
}

NVEncFilterNnedi::~NVEncFilterNnedi() {
    close();
}

NVENCSTATUS NVEncFilterNnedi::checkParam(const std::shared_ptr<NVEncFilterParamNnedi> pNnediParam) {
    if (pNnediParam->frameOut.height <= 0 || pNnediParam->frameOut.width <= 0) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid frame size.\n"));
        return NV_ENC_ERR_INVALID_PARAM;
    }
    if (pNnediParam->nnedi.field <= VPP_NNEDI_FIELD_UNKNOWN || VPP_NNEDI_FIELD_MAX <= pNnediParam->nnedi.field) {
        AddMessage(RGY_LOG_ERROR, _T("invalid value for param \"field\": %d\n"), pNnediParam->nnedi.field);
        return NV_ENC_ERR_INVALID_PARAM;
    }
    if (pNnediParam->nnedi.nns < 16 || 256 < pNnediParam->nnedi.nns) {
        pNnediParam->nnedi.nns = clamp(pNnediParam->nnedi.nns, 16, 256);
        AddMessage(RGY_LOG_WARN, _T("nns should be in range of %d - %d.\n"), 16, 256);
    }
    if (pNnediParam->nnedi.nsize <= VPP_NNEDI_NSIZE_UNKNOWN || VPP_NNEDI_NSIZE_MAX <= pNnediParam->nnedi.nsize) {
        AddMessage(RGY_LOG_ERROR, _T("invalid value for param \"nsize\": %d\n"), pNnediParam->nnedi.nsize);
        return NV_ENC_ERR_INVALID_PARAM;
    }
    if (pNnediParam->nnedi.quality <= VPP_NNEDI_QUALITY_UNKNOWN || VPP_NNEDI_QUALITY_MAX <= pNnediParam->nnedi.quality) {
        AddMessage(RGY_LOG_ERROR, _T("invalid value for param \"quality\": %d\n"), pNnediParam->nnedi.quality);
        return NV_ENC_ERR_INVALID_PARAM;
    }
    if (pNnediParam->nnedi.pre_screen < VPP_NNEDI_PRE_SCREEN_NONE || VPP_NNEDI_PRE_SCREEN_MAX <= pNnediParam->nnedi.pre_screen) {
        AddMessage(RGY_LOG_ERROR, _T("invalid value for param \"pre_screen\": %d\n"), pNnediParam->nnedi.pre_screen);
        return NV_ENC_ERR_INVALID_PARAM;
    }
    if (pNnediParam->nnedi.precision < VPP_NNEDI_PRECISION_UNKNOWN || VPP_NNEDI_PRECISION_MAX <= pNnediParam->nnedi.precision) {
        AddMessage(RGY_LOG_ERROR, _T("invalid value for param \"prec\": %d\n"), pNnediParam->nnedi.precision);
        return NV_ENC_ERR_INVALID_PARAM;
    }
#if !ENABLE_CUDA_FP16_HOST
    if (pNnediParam->nnedi.precision == VPP_NNEDI_PRECISION_FP16) {
        AddMessage(RGY_LOG_WARN, _T("prec=fp16 not compiled in this build, switching to fp32.\n"));
        pNnediParam->nnedi.precision = VPP_NNEDI_PRECISION_FP32;
    }
#endif
    return NV_ENC_SUCCESS;
}

std::vector<float> NVEncFilterNnedi::readWeights(const tstring& weightFile) {
    std::vector<float> weights;
    const uint32_t expectedFileSize = 13574928u;
    uint64_t weightFileSize = 0;
    if (weightFile.length() == 0) {
        HMODULE hModule = GetModuleHandle(NULL);
        HRSRC hResource = NULL;
        HGLOBAL hResourceData = NULL;
        const char *pDataPtr = NULL;
        if (NULL == hModule) {
            AddMessage(RGY_LOG_ERROR, _T("Failed to get module handle.\n"));
        } else if (NULL == (hResource = FindResource(hModule, _T("NNEDI_WEIGHTBIN"), _T("EXE_DATA")))) {
            AddMessage(RGY_LOG_ERROR, _T("Failed to get resource handle for \"NNEDI_WEIGHTBIN\".\n"));
        } else if (NULL == (hResourceData = LoadResource(hModule, hResource))) {
            AddMessage(RGY_LOG_ERROR, _T("Failed to load resource \"NNEDI_WEIGHTBIN\".\n"));
        } else if (NULL == (pDataPtr = (const char *)LockResource(hResourceData))) {
            AddMessage(RGY_LOG_ERROR, _T("Failed to lock resource \"NNEDI_WEIGHTBIN\".\n"));
        } else if (expectedFileSize != (weightFileSize = SizeofResource(hModule, hResource))) {
            AddMessage(RGY_LOG_ERROR, _T("Weights data has unexpected size %u [expected: %u].\n"),
                weightFile.c_str(), (uint32_t)weightFileSize, expectedFileSize);
        } else {
            weights.resize(weightFileSize);
            memcpy(weights.data(), pDataPtr, weightFileSize);
        }
    } else {
        if (!PathFileExists(weightFile.c_str())) {
            AddMessage(RGY_LOG_ERROR, _T("weight file \"%s\" does not exist.\n"), weightFile.c_str());
        } else if (!rgy_get_filesize(weightFile.c_str(), &weightFileSize)) {
            AddMessage(RGY_LOG_ERROR, _T("Failed to get filesize of weight file \"%s\".\n"), weightFile.c_str());
        } else if (weightFileSize != expectedFileSize) {
            AddMessage(RGY_LOG_ERROR, _T("Weights file \"%s\" has unexpected file size %u [expected: %u].\n"),
                weightFile.c_str(), (uint32_t)weightFileSize, expectedFileSize);
        } else {
            weights.resize(weightFileSize);
            std::ifstream fin(weightFile, std::ios::in | std::ios::binary);
            if (!fin.good()) {
                AddMessage(RGY_LOG_ERROR, _T("Failed to open weights file \"%s\".\n"), weightFile.c_str());
            } else if (fin.read((char *)weights.data(), weightFileSize).gcount() != (int64_t)weightFileSize) {
                AddMessage(RGY_LOG_ERROR, _T("Failed to read weights file \"%s\".\n"), weightFile.c_str());
            }
            fin.close();
        }
    }
    return std::move(weights);
}

NVENCSTATUS NVEncFilterNnedi::initParams(const std::shared_ptr<NVEncFilterParamNnedi> pNnediParam) {
    std::vector<float> weights = readWeights(pNnediParam->nnedi.weightfile);
    if (weights.size() == 0) {
        return NV_ENC_ERR_INVALID_PARAM;
    }
    if (pNnediParam->nnedi.precision == VPP_NNEDI_PRECISION_AUTO) {
        pNnediParam->nnedi.precision =
#if ENABLE_CUDA_FP16_HOST
            ((pNnediParam->compute_cpability.first == 6 && pNnediParam->compute_cpability.second == 0)
                || pNnediParam->compute_cpability.first >= 7)
            ? VPP_NNEDI_PRECISION_FP16 : VPP_NNEDI_PRECISION_FP32;
#else
            VPP_NNEDI_PRECISION_FP32;
#endif
    }

    const int weight1size = pNnediParam->nnedi.nns * 2 * (sizeNX[pNnediParam->nnedi.nsize] * sizeNY[pNnediParam->nnedi.nsize] + 1);
    const int sizeofweight1 = (pNnediParam->nnedi.precision == VPP_NNEDI_PRECISION_FP32) ? 4 : 2;
    int weight1size_tsize = 0;
    int weight1size_offset = 0;
    for (int j = 0; j < (int)_countof(sizeNN); j++) {
        for (int i = 0; i < (int)_countof(sizeNX); i++) {
            if (i == pNnediParam->nnedi.nsize
                && j == get_cx_index(list_vpp_nnedi_nns, pNnediParam->nnedi.nns)) {
                weight1size_offset = weight1size_tsize;
            }
            weight1size_tsize += sizeNN[j] * (sizeNX[i] * sizeNY[i] + 1) * 4;
        }
    }

    std::vector<float> weight0f;
    std::array<std::vector<char>, 2> weight1;

    for (size_t i = 0; i < weight1.size(); i++) {
        weight1[i].resize(weight1size * sizeofweight1, 0);
    }

    if (pNnediParam->nnedi.pre_screen >= VPP_NNEDI_PRE_SCREEN_NEW) {
        auto index = [](int j, int k) {
            return ((k >> 3) << 5) + ((j & 3) << 3) + (k & 7);
        };

        const auto ptr_w = weights.data() + weight0size + weight0sizenew * (pNnediParam->nnedi.pre_screen - VPP_NNEDI_PRE_SCREEN_NEW);
        double avg[4] = { 0.0, 0.0, 0.0, 0.0 };
        for (int j = 0; j < 4; j++) {
            double sum = 0.0;
            for (int k = 0; k < 64; k++) {
                sum += ptr_w[index(j, k)];
            }
            avg[j] = sum * (1.0 / 64.0);
        }

        weight0f.resize(weight0sizenew);
#if 0
        for (int j = 0; j < 4; j++) {
            double mval = 0.0;
            for (int k = 0; k < 64; k++) {
                mval = std::max(mval, std::abs((ptr_w[index(j, k)] - avg[j]) * (1.0 / 127.5)));
            }
            const double scale = 32767.0 / mval;
            for (int k = 0; k < 64; k++) {
                m_weight0s[index(j, k)] = (int16_t)(((ptr_w[index(j, k)] - avg[j]) * (1.0 / 127.5)) * scale + 0.5);
            }
            weight0f.push_back(mval * (1.0 / 32767.0));
        }
        for (int i = 0; i < weight0sizenew - 4 * 64; i++) {
            weight0f[i+4] = ptr_w[i + 4 * 64];
        }
#else
        const double halfinv = 1.0 / (((1 << 8) - 1) * 0.5);
        for (int j = 0; j < 4; j++) {
            for (int k = 0; k < 64; k++) {
                //weight0f[index(j, k)] = (float)((ptr_w[index(j, k)] - avg[j]) * halfinv);
                weight0f[j*64+k] = (float)((ptr_w[index(j, k)] - avg[j]) * halfinv);
            }
        }
        for (int i = 0; i < 4; i++) {
            weight0f[4*64+i] = ptr_w[4*64+i];
        }
        for (int j = 0; j < 4; j++) {
            for (int k = 0; k < 4; k++) {
                weight0f[4*65+j*4+k] = ptr_w[4*65+ j + k*4]; //転置
            }
        }
        for (int i = 0; i < 4; i++) {
            weight0f[4*65+4*4+i] = ptr_w[4*65+4*4+i];
        }
#endif
    } else {
        const auto ptr_w = weights.data();
        double avg[4] = { 0.0, 0.0, 0.0, 0.0 };
        for (int j = 0; j < 4; j++) {
            double sum = 0.0;
            for (int k = 0; k < 48; k++) {
                sum += ptr_w[j * 48 + k];
            }
            avg[j] = sum * (1.0 / 48.0);
        }
        weight0f.resize(weight0size);
        const double halfinv = 1.0 / (((1 << 8) - 1) * 0.5);
        for (int j = 0; j < 4; j++) {
            for (int k = 0; k < 48; k++) {
                weight0f[j * 48 + k] = (float)((ptr_w[j * 48 + k] - avg[j]) * halfinv);
            }
        }
        for (int i = 4 * 48; i < weight0size; i++) {
            weight0f[i] = ptr_w[i];
        }
    }

    for (int i = 0; i < 2; i++) {
        const float *ptrW = weights.data() + weight0size + weight0sizenew * 3 + weight1size_tsize * pNnediParam->nnedi.errortype + weight1size_offset + i * weight1size;
        if (pNnediParam->nnedi.precision == VPP_NNEDI_PRECISION_FP32) {
            setWeight1<float>((float *)weight1[i].data(), ptrW, pNnediParam);
        } else {
#if ENABLE_CUDA_FP16_HOST
            setWeight1<__half>((__half *)weight1[i].data(), ptrW, pNnediParam);
#endif //#if ENABLE_CUDA_FP16_HOST
        }
    }
    m_weight0 = CUMemBuf(weight0f.size() * sizeof(weight0f[0]));
    m_weight0.alloc();
    hipMemcpy(m_weight0.ptr, weight0f.data(), m_weight0.nSize, hipMemcpyHostToDevice);
    for (size_t i = 0; i < weight1.size(); i++) {
        m_weight1[i] = CUMemBuf(weight1[i].size());
        m_weight1[i].alloc();
        hipMemcpy(m_weight1[i].ptr, weight1[i].data(), m_weight1[i].nSize, hipMemcpyHostToDevice);
    }
    return NV_ENC_SUCCESS;
}

template<typename TypeWeight> TypeWeight toWeight(float f, float scale);
template<> float toWeight<float>(float f, float scale) { return f; }
#if ENABLE_CUDA_FP16_HOST
template<> __half toWeight<__half>(float f, float scale) { return __float2half_rn(f * scale); }
#endif

template<typename TypeWeight>
void NVEncFilterNnedi::setWeight1(TypeWeight *ptrDst, const float *ptrW, const std::shared_ptr<NVEncFilterParamNnedi> pNnediParam) {
    const int sizeNXY = sizeNX[pNnediParam->nnedi.nsize] * sizeNY[pNnediParam->nnedi.nsize];

    std::vector<double> mean0(pNnediParam->nnedi.nns * 2, 0.0);
    for (int j = 0; j < pNnediParam->nnedi.nns * 2; j++) {
        const float *ptr = ptrW + j * sizeNXY;
        mean0[j] = std::accumulate(ptr, ptr + sizeNXY, 0.0) / (double)sizeNXY;
    }

    const double inv_nns = 1.0 / (double)pNnediParam->nnedi.nns;
    std::vector<double> mean1(sizeNXY, 0.0);
    for (int j = 0; j < pNnediParam->nnedi.nns; j++) {
        for (int k = 0; k < sizeNXY; k++) {
            mean1[k] += (ptrW[j * sizeNXY + k] - mean0[j]) * inv_nns;
        }
    }

    const float *ptr = ptrW + pNnediParam->nnedi.nns * 2 * sizeNXY;
    const double mean2 = std::accumulate(ptr, ptr + pNnediParam->nnedi.nns, 0.0) * inv_nns;

    vector<float> buf(pNnediParam->nnedi.nns * 2 * sizeNXY);
    float max0 = 0.0f, max1 = 0.0f;
    for (int j = 0; j < pNnediParam->nnedi.nns * 2; j++) {
        for (int k = 0; k < sizeNXY; k++) {
            buf[j * sizeNXY + k] = (float)(ptrW[j * sizeNXY + k] - mean0[j] - (j < pNnediParam->nnedi.nns ? mean1[k] : 0.0));
            if (j < pNnediParam->nnedi.nns) {
                max0 = std::max(max0, buf[j * sizeNXY + k]);
            } else {
                max1 = std::max(max1, buf[j * sizeNXY + k]);
            }
        }
        ptrDst[pNnediParam->nnedi.nns * 2 * sizeNXY + j] = toWeight<TypeWeight>(ptrW[pNnediParam->nnedi.nns * 2 * sizeNXY + j] - (float)(j < pNnediParam->nnedi.nns ? mean2 : 0.0), 1.0f);
    }
    for (int j = 0; j < pNnediParam->nnedi.nns * 2; j++) {
        float scale = 1.0f; // std::min(1.0f, (j < pNnediParam->nnedi.nns) ? 4096.0f / max0 : 4096.0f / max1);
        for (int k = 0; k < sizeNXY; k++) {
            ptrDst[j * sizeNXY + k] = toWeight<TypeWeight>(buf[j * sizeNXY + k], scale);
        }
    }
    ptrDst[pNnediParam->nnedi.nns * 2 * (sizeNXY + 1) + 0] = toWeight<TypeWeight>(1.0f /*std::max(1.0f, max0 / 4096.0f)*/, 1.0f);
    ptrDst[pNnediParam->nnedi.nns * 2 * (sizeNXY + 1) + 1] = toWeight<TypeWeight>(1.0f /*std::max(1.0f, max1 / 4096.0f)*/, 1.0f);

#if ENABLE_DP1_WEIGHT_ARRAY_OPT
    //最適化のため、本来の並びを変更する
    //[2][nns][nnxy] -> [nns/WEIGHT_LOOP][nnxy][WEIGHT_LOOP][2]
    vector<TypeWeight> tmp(pNnediParam->nnedi.nns * 2 * (sizeNXY + 1));
    memcpy(tmp.data(), ptrDst, sizeof(tmp[0]) * tmp.size());
    for (int j = 0; j < pNnediParam->nnedi.nns * 2; j++) {
        for (int k = 0; k < sizeNXY; k++) {
            const int j1 = j  / pNnediParam->nnedi.nns;
            const int j2 = j  % pNnediParam->nnedi.nns;
            const int j3 = j2 / WEIGHT_LOOP;
            const int w  = j2 % WEIGHT_LOOP;
            ptrDst[((j3 * sizeNXY + k) * WEIGHT_LOOP + w) * 2 + j1] = tmp[j * sizeNXY + k];
        }
    }
    ptrDst += pNnediParam->nnedi.nns * 2 * sizeNXY;
    auto tmp2 = tmp.data() + pNnediParam->nnedi.nns * 2 * sizeNXY;
    for (int j = 0; j < pNnediParam->nnedi.nns; j++) {
        ptrDst[j * 2 + 0] = tmp2[j];
        ptrDst[j * 2 + 1] = tmp2[pNnediParam->nnedi.nns + j];
    }
#endif
}

NVENCSTATUS NVEncFilterNnedi::init(shared_ptr<NVEncFilterParam> pParam, shared_ptr<RGYLog> pPrintMes) {
    NVENCSTATUS sts = NV_ENC_SUCCESS;
    m_pPrintMes = pPrintMes;
    auto pNnediParam = std::dynamic_pointer_cast<NVEncFilterParamNnedi>(pParam);
    if (!pNnediParam) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid parameter type.\n"));
        return NV_ENC_ERR_INVALID_PARAM;
    }
    //パラメータチェック
    if ((sts = checkParam(pNnediParam)) != NV_ENC_SUCCESS) {
        return sts;
    }

    auto cudaerr = AllocFrameBuf(pNnediParam->frameOut, pNnediParam->nnedi.isbob() ? 2 : 1);
    if (cudaerr != hipSuccess) {
        AddMessage(RGY_LOG_ERROR, _T("failed to allocate memory: %s.\n"), char_to_tstring(hipGetErrorName(cudaerr)).c_str());
        return NV_ENC_ERR_OUT_OF_MEMORY;
    }
    pNnediParam->frameOut.pitch = m_pFrameBuf[0]->frame.pitch;

    auto pNnediParamPrev = std::dynamic_pointer_cast<NVEncFilterParamNnedi>(m_pParam);
    if (!pNnediParamPrev
        || pNnediParamPrev->nnedi != pNnediParam->nnedi) {
        if ((sts = initParams(pNnediParam)) != NV_ENC_SUCCESS) {
            return sts;
        }
    }
    if (pNnediParam->nnedi.isbob()) {
        pParam->baseFps *= 2;
        m_nPathThrough &= (~(FILTER_PATHTHROUGH_TIMESTAMP));
    }

    m_sFilterInfo = strsprintf(
        _T("nnedi: field %s, nns %d, nsize %s, quality %s, prec %s\n")
        _T("                       pre_screen %s, errortype %s, weight \"%s\""),
        get_cx_desc(list_vpp_nnedi_field, pNnediParam->nnedi.field),
        pNnediParam->nnedi.nns,
        get_cx_desc(list_vpp_nnedi_nsize, pNnediParam->nnedi.nsize),
        get_cx_desc(list_vpp_nnedi_quality, pNnediParam->nnedi.quality),
        get_cx_desc(list_vpp_nnedi_prec, pNnediParam->nnedi.precision),
        get_cx_desc(list_vpp_nnedi_pre_screen, pNnediParam->nnedi.pre_screen),
        get_cx_desc(list_vpp_nnedi_error_type, pNnediParam->nnedi.errortype),
        ((pNnediParam->nnedi.weightfile.length()) ? pNnediParam->nnedi.weightfile.c_str() : _T("internal")));

    //コピーを保存
    m_pParam = pNnediParam;
    return sts;
}

NVENCSTATUS NVEncFilterNnedi::run_filter(const FrameInfo *pInputFrame, FrameInfo **ppOutputFrames, int *pOutputFrameNum) {
    NVENCSTATUS sts = NV_ENC_SUCCESS;
    if (pInputFrame->ptr == nullptr) {
        return sts;
    }
    auto pNnediParam = std::dynamic_pointer_cast<NVEncFilterParamNnedi>(m_pParam);
    if (!pNnediParam) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid parameter type.\n"));
        return NV_ENC_ERR_INVALID_PARAM;
    }

    *pOutputFrameNum = 1;
    if (ppOutputFrames[0] == nullptr) {
        auto pOutFrame = m_pFrameBuf[m_nFrameIdx].get();
        ppOutputFrames[0] = &pOutFrame->frame;
        ppOutputFrames[0]->picstruct = pInputFrame->picstruct;
        m_nFrameIdx = (m_nFrameIdx + 1) % m_pFrameBuf.size();
        if (pNnediParam->nnedi.isbob()) {
            pOutFrame = m_pFrameBuf[m_nFrameIdx].get();
            ppOutputFrames[1] = &pOutFrame->frame;
            ppOutputFrames[1]->picstruct = pInputFrame->picstruct;
            m_nFrameIdx = (m_nFrameIdx + 1) % m_pFrameBuf.size();
            *pOutputFrameNum = 2;
        }
    }

    const auto memcpyKind = getCudaMemcpyKind(pInputFrame->deivce_mem, ppOutputFrames[0]->deivce_mem);
    if (memcpyKind != hipMemcpyDeviceToDevice) {
        AddMessage(RGY_LOG_ERROR, _T("only supported on device memory.\n"));
        return NV_ENC_ERR_UNSUPPORTED_PARAM;
    }
    if (m_pParam->frameOut.csp != m_pParam->frameIn.csp) {
        AddMessage(RGY_LOG_ERROR, _T("csp does not match.\n"));
        return NV_ENC_ERR_UNSUPPORTED_PARAM;
    }

    NnediTargetField targetField = NNEDI_GEN_FIELD_UNKNOWN;
    if (   pNnediParam->nnedi.field == VPP_NNEDI_FIELD_USE_AUTO
        || pNnediParam->nnedi.field == VPP_NNEDI_FIELD_BOB_AUTO) {
        if ((pInputFrame->picstruct & RGY_PICSTRUCT_INTERLACED) == 0) {
            const auto inputFrameInfoEx = getFrameInfoExtra(pInputFrame);
            hipMemcpy2DAsync(
                ppOutputFrames[0]->ptr,
                ppOutputFrames[0]->pitch,
                pInputFrame->ptr,
                pInputFrame->pitch,
                inputFrameInfoEx.width_byte,
                inputFrameInfoEx.height_total,
                memcpyKind
            );
            return NV_ENC_SUCCESS;
        } else if (pInputFrame->picstruct & RGY_PICSTRUCT_FRAME_TFF) {
            targetField = NNEDI_GEN_FIELD_BOTTOM;
        } else if (pInputFrame->picstruct & RGY_PICSTRUCT_FRAME_BFF) {
            targetField = NNEDI_GEN_FIELD_TOP;
        }
    } else if (pNnediParam->nnedi.field == VPP_NNEDI_FIELD_USE_TOP
        || pNnediParam->nnedi.field == VPP_NNEDI_FIELD_BOB_TOP_BOTTOM) {
        targetField = NNEDI_GEN_FIELD_BOTTOM;
    } else if (pNnediParam->nnedi.field == VPP_NNEDI_FIELD_USE_BOTTOM
        || pNnediParam->nnedi.field == VPP_NNEDI_FIELD_BOB_BOTTOM_TOP) {
        targetField = NNEDI_GEN_FIELD_TOP;
    } else {
        AddMessage(RGY_LOG_ERROR, _T("Not implemented yet.\n"));
        return NV_ENC_ERR_INVALID_PARAM;
    }

    static const std::map<RGY_CSP, decltype(proc_frame<uint8_t, uchar4, 8, float, float>)*> func_list_fp32 ={
        { RGY_CSP_YV12,      proc_frame<uint8_t,  uchar4,   8, float, float> },
        { RGY_CSP_YV12_16,   proc_frame<uint16_t, ushort4, 16, float, float> },
        { RGY_CSP_YUV444,    proc_frame<uint8_t,  uchar4,   8, float, float> },
        { RGY_CSP_YUV444_16, proc_frame<uint16_t, ushort4, 16, float, float> }
    };
#if ENABLE_CUDA_FP16_HOST
    static const std::map<RGY_CSP, decltype(proc_frame<uint8_t, uchar4, 8, __half2, __half2>)*> func_list_fp16 ={
        { RGY_CSP_YV12,      proc_frame<uint8_t,  uchar4,   8, __half2, __half2> },
        { RGY_CSP_YV12_16,   proc_frame<uint16_t, ushort4, 16, __half2, __half2> },
        { RGY_CSP_YUV444,    proc_frame<uint8_t,  uchar4,   8, __half2, __half2> },
        { RGY_CSP_YUV444_16, proc_frame<uint16_t, ushort4, 16, __half2, __half2> }
    };
    const auto& func_list = (pNnediParam->nnedi.precision == VPP_NNEDI_PRECISION_FP32) ? func_list_fp32 : func_list_fp16;
#else
    const auto& func_list = func_list_fp32;
#endif
    if (func_list.count(pInputFrame->csp) == 0) {
        AddMessage(RGY_LOG_ERROR, _T("unsupported csp %s.\n"), RGY_CSP_NAMES[pInputFrame->csp]);
        return NV_ENC_ERR_UNIMPLEMENTED;
    }
    func_list.at(pInputFrame->csp)(ppOutputFrames[0], pInputFrame,
        pNnediParam, targetField,
        m_weight0.ptr,
        m_weight1[0].ptr,
        m_weight1[1].ptr,
        (hipStream_t)0
        );
    auto cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess) {
        AddMessage(RGY_LOG_ERROR, _T("error at nnedi(%s): %s.\n"),
            RGY_CSP_NAMES[pInputFrame->csp],
            char_to_tstring(hipGetErrorString(cudaerr)).c_str());
        return NV_ENC_ERR_INVALID_CALL;
    }
    ppOutputFrames[0]->picstruct = RGY_PICSTRUCT_FRAME;

    if (pNnediParam->nnedi.isbob()) {
        targetField = (targetField == NNEDI_GEN_FIELD_BOTTOM) ? NNEDI_GEN_FIELD_TOP : NNEDI_GEN_FIELD_BOTTOM;
        func_list.at(pInputFrame->csp)(ppOutputFrames[1], pInputFrame,
            pNnediParam, targetField,
            m_weight0.ptr,
            m_weight1[0].ptr,
            m_weight1[1].ptr,
            (hipStream_t)0
            );
        cudaerr = hipGetLastError();
        if (cudaerr != hipSuccess) {
            AddMessage(RGY_LOG_ERROR, _T("error at nnedi(%s): %s.\n"),
                RGY_CSP_NAMES[pInputFrame->csp],
                char_to_tstring(hipGetErrorString(cudaerr)).c_str());
            return NV_ENC_ERR_INVALID_CALL;
        }
        ppOutputFrames[1]->picstruct = RGY_PICSTRUCT_FRAME;
        ppOutputFrames[0]->timestamp = pInputFrame->timestamp;
        ppOutputFrames[0]->duration = (pInputFrame->duration + 1) / 2;
        ppOutputFrames[1]->timestamp = ppOutputFrames[0]->timestamp + ppOutputFrames[0]->duration;
        ppOutputFrames[1]->duration = pInputFrame->duration - ppOutputFrames[0]->duration;
    }
    return sts;
}

void NVEncFilterNnedi::close() {
    m_pFrameBuf.clear();
}
