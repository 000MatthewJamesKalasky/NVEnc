#include "hip/hip_runtime.h"
﻿// -----------------------------------------------------------------------------------------
// NVEnc by rigaya
// -----------------------------------------------------------------------------------------
//
// The MIT License
//
// Copyright (c) 2014-2016 rigaya
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//
// ------------------------------------------------------------------------------------------

#include <map>
#include <array>
#include <cstdint>
#include "NVEncFilter.h"
#include "hip/hip_runtime.h"
#include ""

union RGY_CSP_2 {
    struct {
        RGY_CSP a, b;
    } csp;
    uint64_t i;

    RGY_CSP_2() {

    };
    RGY_CSP_2(RGY_CSP _a, RGY_CSP _b) {
        csp.a = _a;
        csp.b = _b;
    };
};

#define BIT_DEPTH_CONV(x) (TypeOut)((out_bit_depth == in_bit_depth) \
    ? (x) \
    : ((out_bit_depth > in_bit_depth) \
        ? ((x) << (out_bit_depth - in_bit_depth)) \
        : ((x) >> (in_bit_depth - out_bit_depth))))

template<typename TypeOut, int out_bit_depth, typename TypeIn, int in_bit_depth>
__global__ void kernel_crop_nv12_nv12(TypeOut *__restrict__ pDst, const int dstPitch, const int dstWidth, const int dstHeight,
    const TypeIn *__restrict__ pSrc, const int srcPitch, const int srcHeight, const int offsetX, const int offsetY) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < dstWidth && y < dstHeight) {
        //Y
        int idst = y * dstPitch + x;
        int isrc = (y + offsetY) * srcPitch + x + offsetX;
        pDst[idst] = pSrc[isrc];
        if (y < (dstHeight >> 1)) {
            //UV
            idst += dstHeight * dstPitch;
            isrc += (srcHeight - (offsetY >> 1)) * srcPitch;
            pDst[idst] = pSrc[isrc];
        }
    }
}

template<typename TypeOut, int out_bit_depth, typename TypeIn, int in_bit_depth>
__global__ void kernel_crop_y(uint8_t *__restrict__ pDst, const int dstPitch, const int dstWidth, const int dstHeight,
    const uint8_t *__restrict__ pSrc, const int srcPitch, const int offsetX, const int offsetY) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < dstWidth && y < dstHeight) {
        //Y
        int idst = y * dstPitch + x * sizeof(TypeOut);
        int isrc = (y + offsetY) * srcPitch + (x + offsetX) * sizeof(TypeIn);
        const TypeIn *ptr_src = (const TypeIn *)(pSrc + isrc);
        TypeOut *ptr_dst = (TypeOut *)(pDst + idst);
        ptr_dst[0] = BIT_DEPTH_CONV(ptr_src[0]);
    }
}

template<typename TypeOut, int out_bit_depth, typename TypeIn, int in_bit_depth>
void crop_y(FrameInfo *pOutputFrame, const FrameInfo *pInputFrame, const sInputCrop *pCrop) {
    dim3 blockSize(64, 4);
    dim3 gridSize(divCeil(pOutputFrame->width, blockSize.x), divCeil(pOutputFrame->height, blockSize.y));
    kernel_crop_y<TypeOut, out_bit_depth, TypeIn, in_bit_depth><<<gridSize, blockSize>>>(
        (uint8_t *)pOutputFrame->ptr, pOutputFrame->pitch, pOutputFrame->width, pOutputFrame->height,
        (uint8_t *)pInputFrame->ptr, pInputFrame->pitch, pCrop->e.left, pCrop->e.up);
}

template<typename TypeOut, int out_bit_depth, typename TypeIn, int in_bit_depth>
void crop_uv(FrameInfo *pOutputFrame, const FrameInfo *pInputFrame, const sInputCrop *pCrop) {
    dim3 blockSize(64, 4);
    dim3 gridSize(divCeil(pOutputFrame->width, blockSize.x), divCeil(pOutputFrame->height >> 1, blockSize.y));
    kernel_crop_y<TypeOut, out_bit_depth, TypeIn, in_bit_depth><<<gridSize, blockSize>>>(
        (uint8_t *)pOutputFrame->ptr + pOutputFrame->pitch * pOutputFrame->height,
        pOutputFrame->pitch, pOutputFrame->width, pOutputFrame->height >> 1,
        (uint8_t *)pInputFrame->ptr + pInputFrame->pitch * pInputFrame->height,
        pInputFrame->pitch, pCrop->e.left, pCrop->e.up >> 1);
}

NVENCSTATUS NVEncFilterCspCrop::convertYBitDepth(FrameInfo *pOutputFrame, const FrameInfo *pInputFrame) {
#define CONV_DEPTH_TO_FROM(to, from) ((to) << 8 | (from))
    static const std::map<int, void (*)(FrameInfo *pOutputFrame, const FrameInfo *pInputFrame, const sInputCrop *pCrop)> crop_y_list = {
        { CONV_DEPTH_TO_FROM(16,  8), crop_y<uint16_t, 16, uint8_t,   8> },
        { CONV_DEPTH_TO_FROM(14,  8), crop_y<uint16_t, 14, uint8_t,   8> },
        { CONV_DEPTH_TO_FROM(12,  8), crop_y<uint16_t, 12, uint8_t,   8> },
        { CONV_DEPTH_TO_FROM(10,  8), crop_y<uint16_t, 10, uint8_t,   8> },
        { CONV_DEPTH_TO_FROM( 9,  8), crop_y<uint16_t,  9, uint8_t,   8> },
        { CONV_DEPTH_TO_FROM( 8, 16), crop_y<uint8_t,   8, uint16_t, 16> },
        { CONV_DEPTH_TO_FROM( 8, 14), crop_y<uint8_t,   8, uint16_t, 14> },
        { CONV_DEPTH_TO_FROM( 8, 12), crop_y<uint8_t,   8, uint16_t, 12> },
        { CONV_DEPTH_TO_FROM( 8, 10), crop_y<uint8_t,   8, uint16_t, 10> },
        { CONV_DEPTH_TO_FROM( 8,  9), crop_y<uint8_t,   8, uint16_t,  9> },
        { CONV_DEPTH_TO_FROM(14, 16), crop_y<uint16_t, 14, uint16_t, 16> },
        { CONV_DEPTH_TO_FROM(12, 16), crop_y<uint16_t, 12, uint16_t, 16> },
        { CONV_DEPTH_TO_FROM(10, 16), crop_y<uint16_t, 10, uint16_t, 16> },
        { CONV_DEPTH_TO_FROM( 9, 16), crop_y<uint16_t,  9, uint16_t, 16> },
    };
    const auto bit_depth_conv = CONV_DEPTH_TO_FROM(RGY_CSP_BIT_DEPTH[pOutputFrame->csp], RGY_CSP_BIT_DEPTH[pInputFrame->csp]);
    if (crop_y_list.count(bit_depth_conv) == 0) {
        AddMessage(RGY_LOG_ERROR, _T("unsupported bit depth conversion: %s -> %s.\n"), RGY_CSP_NAMES[pInputFrame->csp], RGY_CSP_NAMES[pOutputFrame->csp]);
        return NV_ENC_ERR_UNIMPLEMENTED;
    }
#undef CONV_DEPTH_TO_FROM
    auto pCropParam = std::dynamic_pointer_cast<NVEncFilterParamCrop>(m_pParam);
    if (!pCropParam) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid parameter type.\n"));
        return NV_ENC_ERR_INVALID_PARAM;
    }
    crop_y_list.at(bit_depth_conv)(pOutputFrame, pInputFrame, &pCropParam->crop);
    auto cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess) {
        AddMessage(RGY_LOG_ERROR, _T("error at convertYBitDepth(%s -> %s): %s.\n"),
            RGY_CSP_NAMES[pInputFrame->csp], RGY_CSP_NAMES[pOutputFrame->csp],
            char_to_tstring(hipGetErrorString(cudaerr)).c_str());
        return NV_ENC_ERR_INVALID_CALL;
    }
    return NV_ENC_SUCCESS;
}

template<typename TypeOut, int out_bit_depth, typename TypeIn, int in_bit_depth>
__global__ void kernel_crop_uv_nv12_yv12(uint8_t *__restrict__ pDstU, uint8_t *__restrict__ pDstV,
    const int dstPitch, const int dstWidth, const int dstHeight,
    const uint8_t *__restrict__ pSrc, const int srcPitch, const int offsetX, const int offsetY) {
    int uv_x = blockIdx.x * blockDim.x + threadIdx.x;
    int uv_y = blockIdx.y * blockDim.y + threadIdx.y;
    if (uv_x < (dstWidth >> 1) && uv_y < (dstHeight >> 1)) {
        int idst = uv_y * dstPitch + uv_x * sizeof(TypeOut); //YV12
        int isrc = (uv_y + (offsetY >> 1)) * srcPitch + ((uv_x << 1) + offsetX) * sizeof(TypeIn); //NV12
        const TypeIn *ptr_src = (const TypeIn *)(pSrc  + isrc);
        TypeOut *ptr_dst_u = (TypeOut *)(pDstU + idst);
        TypeOut *ptr_dst_v = (TypeOut *)(pDstV + idst);
        ptr_dst_u[0] = BIT_DEPTH_CONV(ptr_src[0]);
        ptr_dst_v[0] = BIT_DEPTH_CONV(ptr_src[1]);
    }
}

template<typename TypeOut, int out_bit_depth, typename TypeIn, int in_bit_depth>
void crop_uv_nv12_yv12(FrameInfo *pOutputFrame, const FrameInfo *pInputFrame, const sInputCrop *pCrop) {
    dim3 blockSize(32, 4);
    dim3 gridSize(divCeil(pOutputFrame->width >> 1, blockSize.x), divCeil(pOutputFrame->height >> 1, blockSize.y));
    uint8_t *ptrU = (uint8_t *)pOutputFrame->ptr + pOutputFrame->pitch * pOutputFrame->height;
    uint8_t *ptrV = (uint8_t *)pOutputFrame->ptr + pOutputFrame->pitch * pOutputFrame->height * 3 / 2;
    const uint8_t *ptrC = (const uint8_t  *)pInputFrame->ptr + pInputFrame->pitch  * pInputFrame->height;
    kernel_crop_uv_nv12_yv12<TypeOut, out_bit_depth, TypeIn, in_bit_depth><<<gridSize, blockSize>>>(
        ptrU, ptrV, pOutputFrame->pitch, pOutputFrame->width, pOutputFrame->height,
        ptrC, pInputFrame->pitch, pCrop->e.left, pCrop->e.up);
}

template<typename TypeOut, int out_bit_depth, typename TypeIn, int in_bit_depth>
__global__ void kernel_crop_uv_nv12_yuv444_p(uint8_t *__restrict__ pDstU, uint8_t *__restrict__ pDstV,
    const int dstPitch, const int dstWidth, const int dstHeight,
    const uint8_t *__restrict__ pSrc, const int srcPitch, const int srcWidth, const int srcHeight, const int offsetX, const int offsetY) {
    int uv_x = blockIdx.x * blockDim.x + threadIdx.x;
    int uv_y = blockIdx.y * blockDim.y + threadIdx.y;
    if (uv_x < (dstWidth >> 1) && uv_y < (dstHeight >> 1)) {
        int idst = (uv_y << 1) * dstPitch + (uv_x << 1) * sizeof(TypeOut); //YUV444
        int isrc = (uv_y + (offsetY >> 1)) * srcPitch + ((uv_x << 1) + offsetX) * sizeof(TypeIn); //NV12
        const TypeIn *ptr_src = (const TypeIn *)(pSrc  + isrc);
        TypeOut *ptr_dst_u = (TypeOut *)(pDstU + idst);
        TypeOut *ptr_dst_v = (TypeOut *)(pDstV + idst);
        const int y0_offset  = (uv_y > 0) ? -1 * srcPitch : 0;
        const int y2_offset  = (uv_y+1 < (srcHeight >> 1)) ? srcPitch : 0;
        const int next_pixel = (uv_x+1 < (dstWidth >> 1)) ? 2 : 0;
        const int u_y0x0 = ptr_src[y0_offset+0];
        const int v_y0x0 = ptr_src[y0_offset+1];
        const int u_y0x1 = (ptr_src[y0_offset+next_pixel+0] + u_y0x0 + 1) >> 1;
        const int v_y0x1 = (ptr_src[y0_offset+next_pixel+1] + v_y0x0 + 1) >> 1;
        const int u_y1x0 = ptr_src[0];
        const int v_y1x0 = ptr_src[1];
        const int u_y1x1 = (ptr_src[next_pixel+0] + u_y1x0 + 1) >> 1;
        const int v_y1x1 = (ptr_src[next_pixel+1] + v_y1x0 + 1) >> 1;
        const int u_y2x0 = ptr_src[y2_offset+0];
        const int v_y2x0 = ptr_src[y2_offset+1];
        const int u_y2x1 = (ptr_src[y2_offset+next_pixel+0] + u_y2x0 + 1) >> 1;
        const int v_y2x1 = (ptr_src[y2_offset+next_pixel+1] + v_y2x0 + 1) >> 1;

#define BIT_DEPTH_CONV_a3_b1_rsh2(a, b) (TypeOut)((out_bit_depth == in_bit_depth + 2) \
    ? (3 * (a) + (b) + 2) \
    : ((out_bit_depth > in_bit_depth + 2) \
        ? ((3 * (a) + (b) + 2) << (out_bit_depth - in_bit_depth - 2)) \
        : ((3 * (a) + (b) + 2) >> (in_bit_depth + 2 - out_bit_depth))))

        ptr_dst_u[0] = BIT_DEPTH_CONV_a3_b1_rsh2(u_y1x0, u_y0x0);
        ptr_dst_v[0] = BIT_DEPTH_CONV_a3_b1_rsh2(v_y1x0, v_y0x0);
        ptr_dst_u[1] = BIT_DEPTH_CONV_a3_b1_rsh2(u_y1x1, u_y0x1);
        ptr_dst_v[1] = BIT_DEPTH_CONV_a3_b1_rsh2(v_y1x1, v_y0x1);
        ptr_dst_u = (TypeOut *)((uint8_t *)ptr_dst_u + dstPitch);
        ptr_dst_v = (TypeOut *)((uint8_t *)ptr_dst_v + dstPitch);
        ptr_dst_u[0] = BIT_DEPTH_CONV_a3_b1_rsh2(u_y1x0, u_y2x0);
        ptr_dst_v[0] = BIT_DEPTH_CONV_a3_b1_rsh2(v_y1x0, v_y2x0);
        ptr_dst_u[1] = BIT_DEPTH_CONV_a3_b1_rsh2(u_y1x1, u_y2x1);
        ptr_dst_v[1] = BIT_DEPTH_CONV_a3_b1_rsh2(v_y1x1, v_y2x1);
    }
}

template<typename TypeOut, int out_bit_depth, typename TypeIn, int in_bit_depth>
__global__ void kernel_crop_uv_nv12_yuv444_i(uint8_t *__restrict__ pDstU, uint8_t *__restrict__ pDstV,
    const int dstPitch, const int dstWidth, const int dstHeight,
    const uint8_t *__restrict__ pSrc, const int srcPitch, const int srcWidth, const int srcHeight, const int offsetX, const int offsetY) {
    int uv_x = blockIdx.x * blockDim.x + threadIdx.x;
    int uv_y = (blockIdx.y * blockDim.y + threadIdx.y) << 1;
    if (uv_x < (dstWidth >> 1) && uv_y < (dstHeight >> 1)) {
        int idst = (uv_y << 1) * dstPitch + (uv_x << 1) * sizeof(TypeOut); //YUV444
        int isrc = (uv_y + (offsetY >> 1)) * srcPitch + ((uv_x << 1) + offsetX) * sizeof(TypeIn); //NV12
        const TypeIn *ptr_src = (const TypeIn *)(pSrc  + isrc);
        TypeOut *ptr_dst_u = (TypeOut *)(pDstU + idst);
        TypeOut *ptr_dst_v = (TypeOut *)(pDstV + idst);
        const int y0_offset  = (uv_y - 1 > 0) ? -2 * srcPitch : 0;
        const int y1_offset  = (uv_y > 0)     ? -1 * srcPitch : srcPitch;
        const int y3_offset  = (uv_y+1 < (srcHeight >> 1)) ? srcPitch     : y1_offset;
        const int y4_offset  = (uv_y+2 < (srcHeight >> 1)) ? srcPitch * 2 : 0;
        const int y5_offset  = (uv_y+3 < (srcHeight >> 1)) ? srcPitch * 3 : y3_offset;
        const int next_pixel = (uv_x+1 < (dstWidth >> 1)) ? 2 : 0;
        const int u_y0x0 = ptr_src[y0_offset+0];
        const int v_y0x0 = ptr_src[y0_offset+1];
        const int u_y0x1 = (ptr_src[y0_offset+next_pixel+0] + u_y0x0 + 1) >> 1;
        const int v_y0x1 = (ptr_src[y0_offset+next_pixel+1] + v_y0x0 + 1) >> 1;
        const int u_y1x0 = ptr_src[y1_offset+0];
        const int v_y1x0 = ptr_src[y1_offset+1];
        const int u_y1x1 = (ptr_src[y1_offset+next_pixel+0] + u_y1x0 + 1) >> 1;
        const int v_y1x1 = (ptr_src[y1_offset+next_pixel+1] + v_y1x0 + 1) >> 1;
        const int u_y2x0 = ptr_src[0];
        const int v_y2x0 = ptr_src[1];
        const int u_y2x1 = (ptr_src[next_pixel+0] + u_y2x0 + 1) >> 1;
        const int v_y2x1 = (ptr_src[next_pixel+1] + v_y2x0 + 1) >> 1;
        const int u_y3x0 = ptr_src[y3_offset+0];
        const int v_y3x0 = ptr_src[y3_offset+1];
        const int u_y3x1 = (ptr_src[y3_offset+next_pixel+0] + u_y3x0 + 1) >> 1;
        const int v_y3x1 = (ptr_src[y3_offset+next_pixel+1] + v_y3x0 + 1) >> 1;
        const int u_y4x0 = ptr_src[y4_offset+0];
        const int v_y4x0 = ptr_src[y4_offset+1];
        const int u_y4x1 = (ptr_src[y4_offset+next_pixel+0] + u_y4x0 + 1) >> 1;
        const int v_y4x1 = (ptr_src[y4_offset+next_pixel+1] + v_y4x0 + 1) >> 1;
        const int u_y5x0 = ptr_src[y5_offset+0];
        const int v_y5x0 = ptr_src[y5_offset+1];
        const int u_y5x1 = (ptr_src[y5_offset+next_pixel+0] + u_y5x0 + 1) >> 1;
        const int v_y5x1 = (ptr_src[y5_offset+next_pixel+1] + v_y5x0 + 1) >> 1;

#define BIT_DEPTH_CONV_ia_jb_rsh3(i, a, j, b) (TypeOut)((out_bit_depth == in_bit_depth + 3) \
    ? ((i) * (a) + (j) * (b) + 4) \
    : ((out_bit_depth > in_bit_depth + 2) \
        ? (((i) * (a) + (j) * (b) + 4) << (out_bit_depth - in_bit_depth - 3)) \
        : (((i) * (a) + (j) * (b) + 4) >> (in_bit_depth + 3 - out_bit_depth))))

        ptr_dst_u[0] = BIT_DEPTH_CONV_ia_jb_rsh3(1, u_y0x0, 7, u_y2x0);
        ptr_dst_v[0] = BIT_DEPTH_CONV_ia_jb_rsh3(1, v_y0x0, 7, v_y2x0);
        ptr_dst_u[1] = BIT_DEPTH_CONV_ia_jb_rsh3(1, u_y0x1, 7, u_y2x1);
        ptr_dst_v[1] = BIT_DEPTH_CONV_ia_jb_rsh3(1, v_y0x1, 7, v_y2x1);
        ptr_dst_u = (TypeOut *)((uint8_t *)ptr_dst_u + dstPitch);
        ptr_dst_v = (TypeOut *)((uint8_t *)ptr_dst_v + dstPitch);
        ptr_dst_u[0] = BIT_DEPTH_CONV_ia_jb_rsh3(3, u_y1x0, 5, u_y3x0);
        ptr_dst_v[0] = BIT_DEPTH_CONV_ia_jb_rsh3(3, v_y1x0, 5, v_y3x0);
        ptr_dst_u[1] = BIT_DEPTH_CONV_ia_jb_rsh3(3, u_y1x1, 5, u_y3x1);
        ptr_dst_v[1] = BIT_DEPTH_CONV_ia_jb_rsh3(3, v_y1x1, 5, v_y3x1);
        ptr_dst_u = (TypeOut *)((uint8_t *)ptr_dst_u + dstPitch);
        ptr_dst_v = (TypeOut *)((uint8_t *)ptr_dst_v + dstPitch);
        ptr_dst_u[0] = BIT_DEPTH_CONV_ia_jb_rsh3(5, u_y2x0, 3, u_y4x0);
        ptr_dst_v[0] = BIT_DEPTH_CONV_ia_jb_rsh3(5, v_y2x0, 3, v_y4x0);
        ptr_dst_u[1] = BIT_DEPTH_CONV_ia_jb_rsh3(5, u_y2x1, 3, u_y4x1);
        ptr_dst_v[1] = BIT_DEPTH_CONV_ia_jb_rsh3(5, v_y2x1, 3, v_y4x1);
        ptr_dst_u = (TypeOut *)((uint8_t *)ptr_dst_u + dstPitch);
        ptr_dst_v = (TypeOut *)((uint8_t *)ptr_dst_v + dstPitch);
        ptr_dst_u[0] = BIT_DEPTH_CONV_ia_jb_rsh3(7, u_y3x0, 1, u_y5x0);
        ptr_dst_v[0] = BIT_DEPTH_CONV_ia_jb_rsh3(7, v_y3x0, 1, v_y5x0);
        ptr_dst_u[1] = BIT_DEPTH_CONV_ia_jb_rsh3(7, u_y3x1, 1, u_y5x1);
        ptr_dst_v[1] = BIT_DEPTH_CONV_ia_jb_rsh3(7, v_y3x1, 1, v_y5x1);
    }
}

template<typename TypeOut, int out_bit_depth, typename TypeIn, int in_bit_depth>
void crop_uv_nv12_yuv444(FrameInfo *pOutputFrame, const FrameInfo *pInputFrame, const sInputCrop *pCrop) {
    dim3 blockSize(32, 4);
    dim3 gridSize(divCeil(pOutputFrame->width >> 1, blockSize.x), divCeil(pOutputFrame->height >> 1, blockSize.y));
    uint8_t *ptrU = (uint8_t *)pOutputFrame->ptr + pOutputFrame->pitch * pOutputFrame->height;
    uint8_t *ptrV = (uint8_t *)pOutputFrame->ptr + pOutputFrame->pitch * pOutputFrame->height * 2;
    const uint8_t *ptrC = (const uint8_t  *)pInputFrame->ptr + pInputFrame->pitch * pInputFrame->height;
    if (pInputFrame->interlaced) {
        kernel_crop_uv_nv12_yuv444_i<TypeOut, out_bit_depth, TypeIn, in_bit_depth><<<gridSize, blockSize>>>(
            ptrU, ptrV, pOutputFrame->pitch, pOutputFrame->width, pOutputFrame->height,
            ptrC, pInputFrame->pitch, pInputFrame->width, pInputFrame->height, pCrop->e.left, pCrop->e.up);
    } else {
        kernel_crop_uv_nv12_yuv444_p<TypeOut, out_bit_depth, TypeIn, in_bit_depth><<<gridSize, blockSize>>>(
            ptrU, ptrV, pOutputFrame->pitch, pOutputFrame->width, pOutputFrame->height,
            ptrC, pInputFrame->pitch, pInputFrame->width, pInputFrame->height, pCrop->e.left, pCrop->e.up);
    }
}

template<typename TypeOut, int out_bit_depth, typename TypeIn, int in_bit_depth>
__global__ void kernel_crop_uv_yv12_nv12(uint8_t *__restrict__ pDst, const int dstPitch, const int dstWidth, const int dstHeight,
    const uint8_t *__restrict__ pSrcU, const uint8_t *__restrict__ pSrcV, const int srcPitch, const int offsetX, const int offsetY) {
    int uv_x = blockIdx.x * blockDim.x + threadIdx.x;
    int uv_y = blockIdx.y * blockDim.y + threadIdx.y;
    if (uv_x < (dstWidth >> 1) && uv_y < (dstHeight >> 1)) {
        int idst = uv_y * dstPitch + (uv_x << 1) * sizeof(TypeOut); //NV12
        int isrc = (uv_y + (offsetY >> 1)) * srcPitch + (uv_x + (offsetX >> 1)) * sizeof(TypeIn); //YV12
        const TypeIn *ptr_src_u = (const TypeIn *)(pSrcU + isrc);
        const TypeIn *ptr_src_v = (const TypeIn *)(pSrcV + isrc);
        TypeOut *ptr_dst = (TypeOut *)(pDst + idst);
        ptr_dst[0] = BIT_DEPTH_CONV(ptr_src_u[0]);
        ptr_dst[1] = BIT_DEPTH_CONV(ptr_src_v[0]);
    }
}

template<typename TypeOut, int out_bit_depth, typename TypeIn, int in_bit_depth>
void crop_uv_yv12_nv12(FrameInfo *pOutputFrame, const FrameInfo *pInputFrame, const sInputCrop *pCrop) {
    dim3 blockSize(32, 4);
    dim3 gridSize(divCeil(pOutputFrame->width >> 1, blockSize.x), divCeil(pOutputFrame->height >> 1, blockSize.y));
    uint8_t *ptrC = (uint8_t *)pOutputFrame->ptr + pOutputFrame->pitch * pOutputFrame->height;
    const uint8_t *ptrU = (const uint8_t *)pInputFrame->ptr + pInputFrame->pitch * pInputFrame->height;
    const uint8_t *ptrV = (const uint8_t *)pInputFrame->ptr + pInputFrame->pitch * pInputFrame->height * 3 / 2;
    kernel_crop_uv_yv12_nv12<TypeOut, out_bit_depth, TypeIn, in_bit_depth><<<gridSize, blockSize>>>(
        ptrC, pOutputFrame->pitch, pOutputFrame->width, pOutputFrame->height,
        ptrU, ptrV, pInputFrame->pitch, pCrop->e.left, pCrop->e.up);
}

NVENCSTATUS NVEncFilterCspCrop::convertCspFromNV12(FrameInfo *pOutputFrame, const FrameInfo *pInputFrame) {
    auto pCropParam = std::dynamic_pointer_cast<NVEncFilterParamCrop>(m_pParam);
    if (!pCropParam) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid parameter type.\n"));
        return NV_ENC_ERR_INVALID_PARAM;
    }
    const auto frameOutInfoEx = getFrameInfoExtra(pOutputFrame);
    //Y
    if (RGY_CSP_BIT_DEPTH[pInputFrame->csp] == RGY_CSP_BIT_DEPTH[pOutputFrame->csp]) {
        auto cudaerr = hipMemcpy2D((uint8_t *)pOutputFrame->ptr, pOutputFrame->pitch,
            (uint8_t *)pInputFrame->ptr + pCropParam->crop.e.left + pCropParam->crop.e.up * pInputFrame->pitch,
            pInputFrame->pitch,
            frameOutInfoEx.width_byte, pOutputFrame->height, hipMemcpyDeviceToDevice);
        if (cudaerr != hipSuccess) {
            AddMessage(RGY_LOG_ERROR, _T("error at hipMemcpy2D (convertCspFromNV12(%s -> %s)): %s.\n"),
                RGY_CSP_NAMES[pInputFrame->csp], RGY_CSP_NAMES[pOutputFrame->csp], char_to_tstring(hipGetErrorString(cudaerr)).c_str());
            return NV_ENC_ERR_INVALID_CALL;
        }
    } else {
        auto ret = convertYBitDepth(pOutputFrame, pInputFrame);
        if (ret != NV_ENC_SUCCESS) {
            return ret;
        }
    }

    //UV
    static const std::map<uint64_t, void (*)(FrameInfo *pOutputFrame, const FrameInfo *pInputFrame, const sInputCrop *pCrop)> convert_from_nv12_list = {
        { RGY_CSP_2(RGY_CSP_NV12, RGY_CSP_YV12   ).i,   crop_uv_nv12_yv12<uint8_t,   8, uint8_t,   8> },
        { RGY_CSP_2(RGY_CSP_NV12, RGY_CSP_YV12_16).i,   crop_uv_nv12_yv12<uint16_t, 16, uint8_t,   8> },
        { RGY_CSP_2(RGY_CSP_NV12, RGY_CSP_YV12_14).i,   crop_uv_nv12_yv12<uint16_t, 14, uint8_t,   8> },
        { RGY_CSP_2(RGY_CSP_NV12, RGY_CSP_YV12_12).i,   crop_uv_nv12_yv12<uint16_t, 12, uint8_t,   8> },
        { RGY_CSP_2(RGY_CSP_NV12, RGY_CSP_YV12_10).i,   crop_uv_nv12_yv12<uint16_t, 10, uint8_t,   8> },
        { RGY_CSP_2(RGY_CSP_NV12, RGY_CSP_YV12_09).i,   crop_uv_nv12_yv12<uint16_t,  9, uint8_t,   8> },
        { RGY_CSP_2(RGY_CSP_P010, RGY_CSP_YV12_16).i,   crop_uv_nv12_yv12<uint16_t, 16, uint16_t, 16> },
        { RGY_CSP_2(RGY_CSP_P010, RGY_CSP_YV12_14).i,   crop_uv_nv12_yv12<uint16_t, 14, uint16_t, 16> },
        { RGY_CSP_2(RGY_CSP_P010, RGY_CSP_YV12_12).i,   crop_uv_nv12_yv12<uint16_t, 12, uint16_t, 16> },
        { RGY_CSP_2(RGY_CSP_P010, RGY_CSP_YV12_10).i,   crop_uv_nv12_yv12<uint16_t, 10, uint16_t, 16> },
        { RGY_CSP_2(RGY_CSP_P010, RGY_CSP_YV12_09).i,   crop_uv_nv12_yv12<uint16_t,  9, uint16_t, 16> },
        { RGY_CSP_2(RGY_CSP_NV12, RGY_CSP_YUV444   ).i, crop_uv_nv12_yuv444<uint8_t,   8, uint8_t,   8> },
        { RGY_CSP_2(RGY_CSP_NV12, RGY_CSP_YUV444_16).i, crop_uv_nv12_yuv444<uint16_t, 16, uint8_t,   8> },
        { RGY_CSP_2(RGY_CSP_NV12, RGY_CSP_YUV444_14).i, crop_uv_nv12_yuv444<uint16_t, 14, uint8_t,   8> },
        { RGY_CSP_2(RGY_CSP_NV12, RGY_CSP_YUV444_12).i, crop_uv_nv12_yuv444<uint16_t, 12, uint8_t,   8> },
        { RGY_CSP_2(RGY_CSP_NV12, RGY_CSP_YUV444_10).i, crop_uv_nv12_yuv444<uint16_t, 10, uint8_t,   8> },
        { RGY_CSP_2(RGY_CSP_NV12, RGY_CSP_YUV444_09).i, crop_uv_nv12_yuv444<uint16_t,  9, uint8_t,   8> },
        { RGY_CSP_2(RGY_CSP_P010, RGY_CSP_YUV444   ).i, crop_uv_nv12_yuv444<uint8_t,   8, uint16_t, 16> },
        { RGY_CSP_2(RGY_CSP_P010, RGY_CSP_YUV444_16).i, crop_uv_nv12_yuv444<uint16_t, 16, uint16_t, 16> },
        { RGY_CSP_2(RGY_CSP_P010, RGY_CSP_YUV444_14).i, crop_uv_nv12_yuv444<uint16_t, 14, uint16_t, 16> },
        { RGY_CSP_2(RGY_CSP_P010, RGY_CSP_YUV444_12).i, crop_uv_nv12_yuv444<uint16_t, 12, uint16_t, 16> },
        { RGY_CSP_2(RGY_CSP_P010, RGY_CSP_YUV444_10).i, crop_uv_nv12_yuv444<uint16_t, 10, uint16_t, 16> },
        { RGY_CSP_2(RGY_CSP_P010, RGY_CSP_YUV444_09).i, crop_uv_nv12_yuv444<uint16_t,  9, uint16_t, 16> },
        { RGY_CSP_2(RGY_CSP_NV12, RGY_CSP_P010     ).i, crop_uv<uint16_t, 16, uint8_t,   8> },
        { RGY_CSP_2(RGY_CSP_P010, RGY_CSP_NV12     ).i, crop_uv<uint8_t,   8, uint16_t, 16> },
    };
    const auto cspconv = RGY_CSP_2(pInputFrame->csp, pOutputFrame->csp);
    if (convert_from_nv12_list.count(cspconv.i) == 0) {
        AddMessage(RGY_LOG_ERROR, _T("unsupported csp conversion: %s -> %s.\n"), RGY_CSP_NAMES[pInputFrame->csp], RGY_CSP_NAMES[pOutputFrame->csp]);
        return NV_ENC_ERR_UNIMPLEMENTED;
    }
    convert_from_nv12_list.at(cspconv.i)(pOutputFrame, pInputFrame, &pCropParam->crop);
    auto cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess) {
        AddMessage(RGY_LOG_ERROR, _T("error at convert_from_nv12_list(%s -> %s): %s.\n"),
            RGY_CSP_NAMES[pInputFrame->csp], RGY_CSP_NAMES[pOutputFrame->csp],
            char_to_tstring(hipGetErrorString(cudaerr)).c_str());
        return NV_ENC_ERR_INVALID_CALL;
    }
    return NV_ENC_SUCCESS;
}
NVENCSTATUS NVEncFilterCspCrop::convertCspFromYV12(FrameInfo *pOutputFrame, const FrameInfo *pInputFrame) {
    auto pCropParam = std::dynamic_pointer_cast<NVEncFilterParamCrop>(m_pParam);
    if (!pCropParam) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid parameter type.\n"));
        return NV_ENC_ERR_INVALID_PARAM;
    }
    const auto frameOutInfoEx = getFrameInfoExtra(&pCropParam->frameOut);
    //Y
    if (RGY_CSP_BIT_DEPTH[pInputFrame->csp] == RGY_CSP_BIT_DEPTH[pOutputFrame->csp]) {
        auto cudaerr = hipMemcpy2D((uint8_t *)pOutputFrame->ptr, pOutputFrame->pitch,
            (uint8_t *)pInputFrame->ptr + pCropParam->crop.e.left + pCropParam->crop.e.up * pInputFrame->pitch,
            pInputFrame->pitch,
            frameOutInfoEx.width_byte, pCropParam->frameOut.height, hipMemcpyDeviceToDevice);
        if (cudaerr != hipSuccess) {
            AddMessage(RGY_LOG_ERROR, _T("error at hipMemcpy2D (convertCspFromYV12(%s -> %s)): %s.\n"),
                RGY_CSP_NAMES[pInputFrame->csp], RGY_CSP_NAMES[pOutputFrame->csp], char_to_tstring(hipGetErrorString(cudaerr)).c_str());
            return NV_ENC_ERR_INVALID_CALL;
        }
    } else {
        auto ret = convertYBitDepth(pOutputFrame, pInputFrame);
        if (ret != NV_ENC_SUCCESS) {
            return ret;
        }
    }

    //UV
    static const std::map<uint64_t, void (*)(FrameInfo *pOutputFrame, const FrameInfo *pInputFrame, const sInputCrop *pCrop)> crop_uv_yv12_nv12_list = {
        { RGY_CSP_2(RGY_CSP_YV12,    RGY_CSP_NV12).i, crop_uv_yv12_nv12<uint8_t,   8, uint8_t,   8> },
        { RGY_CSP_2(RGY_CSP_YV12_16, RGY_CSP_P010).i, crop_uv_yv12_nv12<uint16_t, 16, uint16_t, 16> },
        { RGY_CSP_2(RGY_CSP_YV12_14, RGY_CSP_P010).i, crop_uv_yv12_nv12<uint16_t, 14, uint16_t, 16> },
        { RGY_CSP_2(RGY_CSP_YV12_12, RGY_CSP_P010).i, crop_uv_yv12_nv12<uint16_t, 12, uint16_t, 16> },
        { RGY_CSP_2(RGY_CSP_YV12_10, RGY_CSP_P010).i, crop_uv_yv12_nv12<uint16_t, 10, uint16_t, 16> },
        { RGY_CSP_2(RGY_CSP_YV12_09, RGY_CSP_P010).i, crop_uv_yv12_nv12<uint16_t,  9, uint16_t, 16> },
    };
    const auto cspconv = RGY_CSP_2(pInputFrame->csp, pOutputFrame->csp);
    if (crop_uv_yv12_nv12_list.count(cspconv.i) == 0) {
        AddMessage(RGY_LOG_ERROR, _T("unsupported csp conversion: %s -> %s.\n"), RGY_CSP_NAMES[pInputFrame->csp], RGY_CSP_NAMES[pOutputFrame->csp]);
        return NV_ENC_ERR_UNIMPLEMENTED;
    }
    crop_uv_yv12_nv12_list.at(cspconv.i)(pOutputFrame, pInputFrame, &pCropParam->crop);
    auto cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess) {
        AddMessage(RGY_LOG_ERROR, _T("error at crop_uv_nv12_yv12_list(%s -> %s): %s.\n"),
            RGY_CSP_NAMES[pInputFrame->csp], RGY_CSP_NAMES[pOutputFrame->csp],
            char_to_tstring(hipGetErrorString(cudaerr)).c_str());
        return NV_ENC_ERR_INVALID_CALL;
    }
    return NV_ENC_SUCCESS;

}
NVENCSTATUS NVEncFilterCspCrop::convertCspFromYUV444(FrameInfo *pOutputFrame, const FrameInfo *pInputFrame) {
    AddMessage(RGY_LOG_ERROR, _T("unsupported csp conversion: %s -> %s.\n"), RGY_CSP_NAMES[pInputFrame->csp], RGY_CSP_NAMES[pOutputFrame->csp]);
    return NV_ENC_ERR_UNIMPLEMENTED;
}

NVEncFilterCspCrop::NVEncFilterCspCrop() {
    m_sFilterName = _T("copy/cspconv/crop");
}

NVEncFilterCspCrop::~NVEncFilterCspCrop() {
    close();
}

NVENCSTATUS NVEncFilterCspCrop::init(shared_ptr<NVEncFilterParam> pParam, shared_ptr<RGYLog> pPrintMes) {
    NVENCSTATUS sts = NV_ENC_SUCCESS;
    m_pPrintMes = pPrintMes;
    auto pCropParam = std::dynamic_pointer_cast<NVEncFilterParamCrop>(pParam);
    if (!pCropParam) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid parameter type.\n"));
        return NV_ENC_ERR_INVALID_PARAM;
    }
    //フィルタ名の調整
    m_sFilterName = _T("");
    if (cropEnabled(pCropParam->crop)) {
        m_sFilterName += _T("crop");
    }
    if (pCropParam->frameOut.csp != pCropParam->frameIn.csp) {
        m_sFilterName += (m_sFilterName.length()) ? _T("/cspconv") : _T("cspconv");
    }
    if (m_sFilterName.length() == 0) {
        m_sFilterName += _T("copy");
    }
    //パラメータチェック
    for (int i = 0; i < _countof(pCropParam->crop.c); i++) {
        if ((pCropParam->crop.c[i] & 1) != 0) {
            AddMessage(RGY_LOG_ERROR, _T("crop should be divided by 2.\n"));
            return NV_ENC_ERR_INVALID_PARAM;
        }
    }
    pCropParam->frameOut.height = pCropParam->frameIn.height - pCropParam->crop.e.bottom - pCropParam->crop.e.up;
    pCropParam->frameOut.width = pCropParam->frameIn.width - pCropParam->crop.e.left - pCropParam->crop.e.right;
    if (pCropParam->frameOut.height <= 0 || pCropParam->frameOut.width <= 0) {
        AddMessage(RGY_LOG_ERROR, _T("crop size is too big.\n"));
        return NV_ENC_ERR_INVALID_PARAM;
    }

    auto cudaerr = AllocFrameBuf(pCropParam->frameOut, 2);
    if (cudaerr != hipSuccess) {
        AddMessage(RGY_LOG_ERROR, _T("failed to allocate memory: %s.\n"), char_to_tstring(hipGetErrorName(cudaerr)).c_str());
        return NV_ENC_ERR_OUT_OF_MEMORY;
    }
    pCropParam->frameOut.pitch = m_pFrameBuf[0]->frame.pitch;

    //フィルタ情報の調整
    m_sFilterInfo = _T("");
    if (cropEnabled(pCropParam->crop)) {
        m_sFilterInfo += strsprintf(_T("crop: %d,%d,%d,%d"), pCropParam->crop.e.left, pCropParam->crop.e.up, pCropParam->crop.e.right, pCropParam->crop.e.bottom);
    }
    if (pCropParam->frameOut.csp != pCropParam->frameIn.csp) {
        m_sFilterInfo += (m_sFilterInfo.length()) ? _T("/cspconv") : _T("cspconv");
        m_sFilterInfo += strsprintf(_T("(%s -> %s)"), RGY_CSP_NAMES[pCropParam->frameIn.csp], RGY_CSP_NAMES[pCropParam->frameOut.csp]);
    }
    if (m_sFilterInfo.length() == 0) {
        m_sFilterInfo += getCudaMemcpyKindStr(pCropParam->frameIn.deivce_mem, pCropParam->frameOut.deivce_mem);
    }

    m_pParam = pCropParam;
    return sts;
}

NVENCSTATUS NVEncFilterCspCrop::run_filter(const FrameInfo *pInputFrame, FrameInfo **ppOutputFrames, int *pOutputFrameNum) {
    NVENCSTATUS sts = NV_ENC_SUCCESS;

    *pOutputFrameNum = 1;
    if (ppOutputFrames[0] == nullptr) {
        auto pOutFrame = m_pFrameBuf[m_nFrameIdx].get();
        ppOutputFrames[0] = &pOutFrame->frame;
        m_nFrameIdx = (m_nFrameIdx + 1) % m_pFrameBuf.size();
    }
    auto pCropParam = std::dynamic_pointer_cast<NVEncFilterParamCrop>(m_pParam);
    if (!pCropParam) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid parameter type.\n"));
        return NV_ENC_ERR_INVALID_PARAM;
    }
    const auto memcpyKind = getCudaMemcpyKind(pInputFrame->deivce_mem, ppOutputFrames[0]->deivce_mem);
    ppOutputFrames[0]->interlaced = pInputFrame->interlaced;
    if (m_pParam->frameOut.csp == m_pParam->frameIn.csp) {
        auto cudaMemcpyErrMes = [&](hipError_t cudaerr, const TCHAR *mes) {
            AddMessage(RGY_LOG_ERROR, _T("error at %s (filter(%s)): %s.\n"),
                mes, RGY_CSP_NAMES[pInputFrame->csp], char_to_tstring(hipGetErrorString(cudaerr)).c_str());
            return NV_ENC_ERR_INVALID_CALL;
        };
#if 1
        const auto frameOutInfoEx = getFrameInfoExtra(ppOutputFrames[0]);
        if (!cropEnabled(pCropParam->crop)) {
            //cropがなければ、一度に転送可能
            auto cudaerr = hipMemcpy2D((uint8_t *)ppOutputFrames[0]->ptr, ppOutputFrames[0]->pitch,
                (uint8_t *)pInputFrame->ptr, pInputFrame->pitch,
                frameOutInfoEx.width_byte, frameOutInfoEx.height_total, memcpyKind);
            if (cudaerr != hipSuccess) {
                cudaMemcpyErrMes(cudaerr, _T("cudaMemcpy2DAll"));
                return NV_ENC_ERR_INVALID_CALL;
            };
        } else {
            if (pCropParam->frameOut.csp == RGY_CSP_NV12) {
                hipError_t cudaerr;
                //Y
                cudaerr = hipMemcpy2D((uint8_t *)ppOutputFrames[0]->ptr, ppOutputFrames[0]->pitch,
                    (uint8_t *)pInputFrame->ptr + pCropParam->crop.e.left + pCropParam->crop.e.up * pInputFrame->pitch,
                    pInputFrame->pitch,
                    frameOutInfoEx.width_byte, pCropParam->frameOut.height, memcpyKind);
                if (cudaerr != hipSuccess) {
                    cudaMemcpyErrMes(cudaerr, _T("cudaMemcpy2D_Y"));
                    return NV_ENC_ERR_INVALID_CALL;
                };
                //UV
                cudaerr = hipMemcpy2D((uint8_t *)ppOutputFrames[0]->ptr + ppOutputFrames[0]->pitch * ppOutputFrames[0]->height, ppOutputFrames[0]->pitch,
                    (uint8_t *)pInputFrame->ptr
                    + pInputFrame->height * pInputFrame->pitch
                    + pCropParam->crop.e.left + (pCropParam->crop.e.up >> 1) * pInputFrame->pitch,
                    pInputFrame->pitch,
                    frameOutInfoEx.width_byte, pCropParam->frameOut.height >> 1, memcpyKind);
                if (cudaerr != hipSuccess) {
                    cudaMemcpyErrMes(cudaerr, _T("cudaMemcpy2D_UV"));
                    return NV_ENC_ERR_INVALID_CALL;
                };
            } else {
                AddMessage(RGY_LOG_ERROR, _T("unsupported output csp with crop.\n"));
                return NV_ENC_ERR_UNIMPLEMENTED;
            }
        }
#else
        if (pCropParam->frameOut.csp == RGY_CSP_NV12) {
            dim3 blockSize(32, 4);
            dim3 gridSize(divCeil(pCropParam->frameOut.width, blockSize.x), divCeil(pCropParam->frameOut.height, blockSize.y));
            kernel_crop_nv12_nv12<uint8_t><<<gridSize, blockSize>>>((uint8_t *)ppOutputFrames[0]->ptr, (uint8_t *)pInputFrame->ptr, pInputFrame->pitch);
        } else {
            AddMessage(RGY_LOG_ERROR, _T("unsupported output csp.\n"));
            return NV_ENC_ERR_UNSUPPORTED_PARAM;
        }
#endif
    } else if (memcpyKind != hipMemcpyDeviceToDevice) {
        AddMessage(RGY_LOG_ERROR, _T("converting csp while copying from host to device is not supported.\n"));
        return NV_ENC_ERR_UNSUPPORTED_PARAM;
    } else {
        //色空間変換
        static const auto supportedCspNV12   = make_array<RGY_CSP>(RGY_CSP_NV12, RGY_CSP_P010);
        static const auto supportedCspYV12   = make_array<RGY_CSP>(RGY_CSP_YV12, RGY_CSP_YV12_09, RGY_CSP_YV12_10, RGY_CSP_YV12_12, RGY_CSP_YV12_14, RGY_CSP_YV12_16);
        static const auto supportedCspYUV444 = make_array<RGY_CSP>(RGY_CSP_YUV444, RGY_CSP_YUV444_09, RGY_CSP_YUV444_10, RGY_CSP_YUV444_12, RGY_CSP_YUV444_14, RGY_CSP_YUV444_16);
        if (std::find(supportedCspNV12.begin(), supportedCspNV12.end(), pCropParam->frameIn.csp) != supportedCspNV12.end()) {
            sts = convertCspFromNV12(ppOutputFrames[0], pInputFrame);
        } else if (std::find(supportedCspYV12.begin(), supportedCspYV12.end(), pCropParam->frameIn.csp) != supportedCspYV12.end()) {
            sts = convertCspFromYV12(ppOutputFrames[0], pInputFrame);
        } else if (std::find(supportedCspYUV444.begin(), supportedCspYUV444.end(), pCropParam->frameIn.csp) != supportedCspYUV444.end()) {
            sts = convertCspFromYUV444(ppOutputFrames[0], pInputFrame);
        } else {
            AddMessage(RGY_LOG_ERROR, _T("converting csp from %s is not supported.\n"), RGY_CSP_NAMES[pCropParam->frameIn.csp]);
            sts = NV_ENC_ERR_UNIMPLEMENTED;
        }
    }
    return sts;
}

void NVEncFilterCspCrop::close() {
    m_pFrameBuf.clear();
}
