#include "hip/hip_runtime.h"
﻿// -----------------------------------------------------------------------------------------
// NVEnc by rigaya
// -----------------------------------------------------------------------------------------
//
// The MIT License
//
// Copyright (c) 2014-2016 rigaya
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//
// ------------------------------------------------------------------------------------------

#include <map>
#include <array>
#include "convert_csp.h"
#include "NVEncFilterDenoiseKnn.h"
#include "NVEncParam.h"
#pragma warning (push)
#pragma warning (disable: 4819)
#include "hip/hip_runtime.h"
#include ""
#pragma warning (pop)
#include "rgy_cuda_util_kernel.h"

static const int KNN_RADIUS_MAX = 5;

template<typename Type, int knn_radius, int bit_depth>
__global__ void kernel_denoise_knn(uint8_t *__restrict__ pDst, const int dstPitch, const int dstWidth, const int dstHeight,
    hipTextureObject_t texSrc, const float strength, const float lerpC, const float weight_threshold, const float lerp_threshold) {
    const float knn_window_area = (float)((2 * knn_radius + 1) * (2 * knn_radius + 1));
    const float inv_knn_window_area = 1.0f / knn_window_area;
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (ix < dstWidth && iy < dstHeight) {
        const float x = (float)ix + 0.5f;
        const float y = (float)iy + 0.5f;

        float fCount = 0.0f;
        float sumWeights = 0.0f;
        float sum = 0.0f;
        float center = (float)tex2D<Type>(texSrc, x, y) * (1.0f / (1<<bit_depth));

        for (float i = -knn_radius; i <= knn_radius; i++) {
            for (float j = -knn_radius; j <= knn_radius; j++) {
                float clrIJ = (float)tex2D<Type>(texSrc, x + j, y + i) * (1.0f / (1<<bit_depth));
                float distanceIJ = (center - clrIJ) * (center - clrIJ);

                float weightIJ = __expf(-(distanceIJ * strength + (i * i + j * j) * inv_knn_window_area));

                sum += clrIJ * weightIJ;

                sumWeights += weightIJ;

                fCount += (weightIJ > weight_threshold) ? inv_knn_window_area : 0;
            }
        }
        float lerpQ = (fCount > lerp_threshold) ? lerpC : 1.0f - lerpC;

        Type *ptr = (Type *)(pDst + iy * dstPitch + ix * sizeof(Type));
        ptr[0] = (Type)(lerpf(sum * __frcp_rn(sumWeights), center, lerpQ) * (1<<bit_depth));
    }
}

template<typename Type, int bit_depth>
void denoise_knn(uint8_t *pDst, const int dstPitch, const int dstWidth, const int dstHeight,
    hipTextureObject_t texSrc, int radius, const float strength, const float lerpC, const float weight_threshold, const float lerp_threshold) {
    dim3 blockSize(64, 16);
    dim3 gridSize(divCeil(dstWidth, blockSize.x), divCeil(dstHeight, blockSize.y));
    switch (radius) {
    case 1:
        kernel_denoise_knn<Type, 1, bit_depth><<<gridSize, blockSize>>>(pDst, dstPitch, dstWidth, dstHeight, texSrc,
            1.0f / (strength * strength), lerpC, weight_threshold, lerp_threshold);
        break;
    case 2:
        kernel_denoise_knn<Type, 2, bit_depth><<<gridSize, blockSize>>>(pDst, dstPitch, dstWidth, dstHeight, texSrc,
            1.0f / (strength * strength), lerpC, weight_threshold, lerp_threshold);
        break;
    case 3:
        kernel_denoise_knn<Type, 3, bit_depth><<<gridSize, blockSize>>>(pDst, dstPitch, dstWidth, dstHeight, texSrc,
            1.0f / (strength * strength), lerpC, weight_threshold, lerp_threshold);
        break;
    case 4:
        kernel_denoise_knn<Type, 4, bit_depth><<<gridSize, blockSize>>>(pDst, dstPitch, dstWidth, dstHeight, texSrc,
            1.0f / (strength * strength), lerpC, weight_threshold, lerp_threshold);
        break;
    case 5:
        //よりレジスタを使うので、ブロック当たりのスレッド数を低減
        blockSize = dim3(32, 16);
        gridSize = dim3(divCeil(dstWidth, blockSize.x), divCeil(dstHeight, blockSize.y));
        kernel_denoise_knn<Type, 5, bit_depth><<<gridSize, blockSize>>>(pDst, dstPitch, dstWidth, dstHeight, texSrc,
            1.0f / (strength * strength), lerpC, weight_threshold, lerp_threshold);
        break;
    default:
        break;
    }
}

template<typename Type, int bit_depth>
static hipError_t denoise_yv12(FrameInfo *pOutputFrame, const FrameInfo *pInputFrame,
    int radius, const float strength, const float lerpC, const float weight_threshold, const float lerp_threshold) {
    //Y
    hipResourceDesc resDescSrc;
    memset(&resDescSrc, 0, sizeof(resDescSrc));
    resDescSrc.resType = hipResourceTypePitch2D;
    resDescSrc.res.pitch2D.devPtr = pInputFrame->ptr;
    resDescSrc.res.pitch2D.pitchInBytes = pInputFrame->pitch;
    resDescSrc.res.pitch2D.width = pInputFrame->width;
    resDescSrc.res.pitch2D.height = pInputFrame->height;
    resDescSrc.res.pitch2D.desc = hipCreateChannelDesc<Type>();

    hipTextureDesc texDescSrc;
    memset(&texDescSrc, 0, sizeof(texDescSrc));
    texDescSrc.addressMode[0]   = hipAddressModeClamp;
    texDescSrc.addressMode[1]   = hipAddressModeClamp;
    texDescSrc.filterMode       = hipFilterModePoint;
    texDescSrc.readMode         = hipReadModeElementType;
    texDescSrc.normalizedCoords = 0;

    hipTextureObject_t texSrc = 0;
    auto cudaerr = hipCreateTextureObject(&texSrc, &resDescSrc, &texDescSrc, nullptr);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    denoise_knn<Type, bit_depth>((uint8_t *)pOutputFrame->ptr,
        pOutputFrame->pitch, pOutputFrame->width, pOutputFrame->height,
        texSrc, radius, strength, lerpC, weight_threshold, lerp_threshold);
    cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    cudaerr = hipDestroyTextureObject(texSrc);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    //U
    resDescSrc.res.pitch2D.devPtr = (uint8_t *)pInputFrame->ptr + pInputFrame->pitch * pInputFrame->height;
    resDescSrc.res.pitch2D.width >>= 1;
    resDescSrc.res.pitch2D.height >>= 1;
    cudaerr = hipCreateTextureObject(&texSrc, &resDescSrc, &texDescSrc, nullptr);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    denoise_knn<Type, bit_depth>((uint8_t *)pOutputFrame->ptr + pOutputFrame->pitch * pOutputFrame->height,
        pOutputFrame->pitch, pOutputFrame->width >> 1, pOutputFrame->height >> 1,
        texSrc, radius, strength, lerpC, weight_threshold, lerp_threshold);
    cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    cudaerr = hipDestroyTextureObject(texSrc);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    //V
    resDescSrc.res.pitch2D.devPtr = (uint8_t *)pInputFrame->ptr + pInputFrame->pitch * pInputFrame->height * 3 / 2;
    cudaerr = hipCreateTextureObject(&texSrc, &resDescSrc, &texDescSrc, nullptr);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    denoise_knn<Type, bit_depth>((uint8_t *)pOutputFrame->ptr + pOutputFrame->pitch * pOutputFrame->height * 3 / 2,
        pOutputFrame->pitch, pOutputFrame->width >> 1, pOutputFrame->height >> 1,
        texSrc, radius, strength, lerpC, weight_threshold, lerp_threshold);
    cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    cudaerr = hipDestroyTextureObject(texSrc);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    return cudaerr;
}

template<typename Type, int bit_depth>
static hipError_t denoise_yuv444(FrameInfo *pOutputFrame, const FrameInfo *pInputFrame,
    int radius, const float strength, const float lerpC, const float weight_threshold, const float lerp_threshold) {
    //Y
    hipResourceDesc resDescSrc;
    memset(&resDescSrc, 0, sizeof(resDescSrc));
    resDescSrc.resType = hipResourceTypePitch2D;
    resDescSrc.res.pitch2D.devPtr = pInputFrame->ptr;
    resDescSrc.res.pitch2D.pitchInBytes = pInputFrame->pitch;
    resDescSrc.res.pitch2D.width = pInputFrame->width;
    resDescSrc.res.pitch2D.height = pInputFrame->height;
    resDescSrc.res.pitch2D.desc = hipCreateChannelDesc<Type>();

    hipTextureDesc texDescSrc;
    memset(&texDescSrc, 0, sizeof(texDescSrc));
    texDescSrc.addressMode[0]   = hipAddressModeClamp;
    texDescSrc.addressMode[1]   = hipAddressModeClamp;
    texDescSrc.filterMode       = hipFilterModePoint;
    texDescSrc.readMode         = hipReadModeElementType;
    texDescSrc.normalizedCoords = 0;

    hipTextureObject_t texSrc = 0;
    auto cudaerr = hipCreateTextureObject(&texSrc, &resDescSrc, &texDescSrc, nullptr);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    denoise_knn<Type, bit_depth>((uint8_t *)pOutputFrame->ptr,
        pOutputFrame->pitch, pOutputFrame->width, pOutputFrame->height,
        texSrc, radius, strength, lerpC, weight_threshold, lerp_threshold);
    cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    cudaerr = hipDestroyTextureObject(texSrc);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    //U
    resDescSrc.res.pitch2D.devPtr = (uint8_t *)pInputFrame->ptr + pInputFrame->pitch * pInputFrame->height;
    cudaerr = hipCreateTextureObject(&texSrc, &resDescSrc, &texDescSrc, nullptr);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    denoise_knn<Type, bit_depth>((uint8_t *)pOutputFrame->ptr + pOutputFrame->pitch * pOutputFrame->height,
        pOutputFrame->pitch, pOutputFrame->width, pOutputFrame->height,
        texSrc, radius, strength, lerpC, weight_threshold, lerp_threshold);
    cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    cudaerr = hipDestroyTextureObject(texSrc);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    //V
    resDescSrc.res.pitch2D.devPtr = (uint8_t *)pInputFrame->ptr + pInputFrame->pitch * pInputFrame->height * 2;
    cudaerr = hipCreateTextureObject(&texSrc, &resDescSrc, &texDescSrc, nullptr);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    denoise_knn<Type, bit_depth>((uint8_t *)pOutputFrame->ptr + pOutputFrame->pitch * pOutputFrame->height * 2,
        pOutputFrame->pitch, pOutputFrame->width, pOutputFrame->height,
        texSrc, radius, strength, lerpC, weight_threshold, lerp_threshold);
    cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    cudaerr = hipDestroyTextureObject(texSrc);
    if (cudaerr != hipSuccess) {
        return cudaerr;
    }
    return cudaerr;
}

NVEncFilterDenoiseKnn::NVEncFilterDenoiseKnn() : m_bInterlacedWarn(false) {
    m_sFilterName = _T("knn");
}

NVEncFilterDenoiseKnn::~NVEncFilterDenoiseKnn() {
    close();
}

RGY_ERR NVEncFilterDenoiseKnn::init(shared_ptr<NVEncFilterParam> pParam, shared_ptr<RGYLog> pPrintMes) {
    RGY_ERR sts = RGY_ERR_NONE;
    m_pPrintMes = pPrintMes;
    auto pKnnParam = std::dynamic_pointer_cast<NVEncFilterParamDenoiseKnn>(pParam);
    if (!pKnnParam) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid parameter type.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    //パラメータチェック
    if (pKnnParam->frameOut.height <= 0 || pKnnParam->frameOut.width <= 0) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid parameter.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    if (pKnnParam->knn.radius <= 0) {
        AddMessage(RGY_LOG_ERROR, _T("radius must be a positive value.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    if (pKnnParam->knn.radius > KNN_RADIUS_MAX) {
        AddMessage(RGY_LOG_ERROR, _T("radius must be <= %d.\n"), KNN_RADIUS_MAX);
        return RGY_ERR_INVALID_PARAM;
    }
    if (pKnnParam->knn.strength < 0.0 || 1.0 < pKnnParam->knn.strength) {
        AddMessage(RGY_LOG_ERROR, _T("strength should be 0.0 - 1.0.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    if (pKnnParam->knn.lerpC < 0.0 || 1.0 < pKnnParam->knn.lerpC) {
        AddMessage(RGY_LOG_ERROR, _T("lerpC should be 0.0 - 1.0.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    if (pKnnParam->knn.lerp_threshold < 0.0 || 1.0 < pKnnParam->knn.lerp_threshold) {
        AddMessage(RGY_LOG_ERROR, _T("th_lerp should be 0.0 - 1.0.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    if (pKnnParam->knn.weight_threshold < 0.0 || 1.0 < pKnnParam->knn.weight_threshold) {
        AddMessage(RGY_LOG_ERROR, _T("th_weight should be 0.0 - 1.0.\n"));
        return RGY_ERR_INVALID_PARAM;
    }

    auto cudaerr = AllocFrameBuf(pKnnParam->frameOut, 1);
    if (cudaerr != hipSuccess) {
        AddMessage(RGY_LOG_ERROR, _T("failed to allocate memory: %s.\n"), char_to_tstring(hipGetErrorName(cudaerr)).c_str());
        return RGY_ERR_MEMORY_ALLOC;
    }
    pKnnParam->frameOut.pitch = m_pFrameBuf[0]->frame.pitch;

    setFilterInfo(pParam->print());
    m_pParam = pParam;
    return sts;
}

tstring NVEncFilterParamDenoiseKnn::print() const {
    return knn.print();
}

RGY_ERR NVEncFilterDenoiseKnn::run_filter(const FrameInfo *pInputFrame, FrameInfo **ppOutputFrames, int *pOutputFrameNum, hipStream_t stream) {
    RGY_ERR sts = RGY_ERR_NONE;

    if (pInputFrame->ptr == nullptr) {
        return sts;
    }

    *pOutputFrameNum = 1;
    if (ppOutputFrames[0] == nullptr) {
        auto pOutFrame = m_pFrameBuf[m_nFrameIdx].get();
        ppOutputFrames[0] = &pOutFrame->frame;
        m_nFrameIdx = (m_nFrameIdx + 1) % m_pFrameBuf.size();
    }
    ppOutputFrames[0]->picstruct = pInputFrame->picstruct;
    if (interlaced(*pInputFrame)) {
        return filter_as_interlaced_pair(pInputFrame, ppOutputFrames[0], hipStreamDefault);
    }
    const auto memcpyKind = getCudaMemcpyKind(pInputFrame->deivce_mem, ppOutputFrames[0]->deivce_mem);
    if (memcpyKind != hipMemcpyDeviceToDevice) {
        AddMessage(RGY_LOG_ERROR, _T("only supported on device memory.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    if (m_pParam->frameOut.csp != m_pParam->frameIn.csp) {
        AddMessage(RGY_LOG_ERROR, _T("csp does not match.\n"));
        return RGY_ERR_INVALID_PARAM;
    }
    auto pKnnParam = std::dynamic_pointer_cast<NVEncFilterParamDenoiseKnn>(m_pParam);
    if (!pKnnParam) {
        AddMessage(RGY_LOG_ERROR, _T("Invalid parameter type.\n"));
        return RGY_ERR_INVALID_PARAM;
    }

    static const std::map<RGY_CSP, decltype(denoise_yv12<uint8_t, 8>)*> denoise_list = {
        { RGY_CSP_YV12,      denoise_yv12<uint8_t,   8> },
        { RGY_CSP_YV12_16,   denoise_yv12<uint16_t, 16> },
        { RGY_CSP_YUV444,    denoise_yuv444<uint8_t,   8> },
        { RGY_CSP_YUV444_16, denoise_yuv444<uint16_t, 16> },
    };
    if (denoise_list.count(pInputFrame->csp) == 0) {
        AddMessage(RGY_LOG_ERROR, _T("unsupported csp %s.\n"), RGY_CSP_NAMES[pInputFrame->csp]);
        return RGY_ERR_UNSUPPORTED;
    }
    denoise_list.at(pInputFrame->csp)(ppOutputFrames[0], pInputFrame, pKnnParam->knn.radius, pKnnParam->knn.strength, pKnnParam->knn.lerpC, pKnnParam->knn.weight_threshold, pKnnParam->knn.lerp_threshold);
    auto cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess) {
        AddMessage(RGY_LOG_ERROR, _T("error at resize(%s): %s.\n"),
            RGY_CSP_NAMES[pInputFrame->csp],
            char_to_tstring(hipGetErrorString(cudaerr)).c_str());
        return RGY_ERR_CUDA;
    }
    return sts;
}

void NVEncFilterDenoiseKnn::close() {
    m_pFrameBuf.clear();
    m_bInterlacedWarn = false;
}
